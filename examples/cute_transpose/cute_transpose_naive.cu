#include "hip/hip_runtime.h"
#include <iostream>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <cute/tensor.hpp>

#include <hip/hip_runtime.h>

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
void check(hipError_t err, char const* func, char const* file, int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK_LAST_CUDA_ERROR() checkLast(__FILE__, __LINE__)
void checkLast(char const* file, int const line)
{
    hipError_t const err{hipGetLastError()};
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

template <class TENSOR_SRC, class TENSOR_DST, class THREAD_LAYOUT>
__global__ void transpose_naive(TENSOR_SRC tensor_src,
                                TENSOR_DST tensor_dst_transposed, THREAD_LAYOUT)
{
    using Element = typename TENSOR_SRC::value_type;

    auto global_tile_src{tensor_src(cute::make_coord(cute::_, cute::_),
                                    blockIdx.y,
                                    blockIdx.x)}; // (TILE_SIZE_Y, TILE_SIZE_X)
    auto global_tile_dst_transposed{
        tensor_dst_transposed(cute::make_coord(cute::_, cute::_), blockIdx.y,
                              blockIdx.x)}; // (TILE_SIZE_Y, TILE_SIZE_X)

    auto thread_tile_src{cute::local_partition(
        global_tile_src, THREAD_LAYOUT{},
        threadIdx.x)}; // (THREAD_VALUE_SIZE_Y, THREAD_VALUE_SIZE_X)
    auto thread_tile_dst_transposed{cute::local_partition(
        global_tile_dst_transposed, THREAD_LAYOUT{},
        threadIdx.x)}; // (THREAD_VALUE_SIZE_Y, THREAD_VALUE_SIZE_X)

    auto register_fragment{cute::make_tensor_like(thread_tile_src)};
    // auto predicate{cute::make_tensor_like<bool>(thread_tile_src)};

    // for (unsigned int i{0}; i < cute::size<0>(thread_tile_src); ++i)
    // {
    //     for (unsigned int j{0}; j < cute::size<1>(thread_tile_src); ++j)
    //     {
    //         predicate(i, j) = cute::get<0>(thread_tile_src(i, j)) < 1021 &&
    //                           cute::get<1>(thread_tile_src(i, j)) < 2049;
    //     }
    // }

    cute::copy(thread_tile_src, register_fragment);
    cute::copy(register_fragment, thread_tile_dst_transposed);
}

template <class T>
void transpose(T const* src, T* dst, unsigned int M, unsigned int N)
{
    for (unsigned int i{0}; i < N; ++i)
    {
        for (unsigned int j{0}; j < M; ++j)
        {
            dst[j * N + i] = src[i * M + j];
        }
    }
}

template <class T>
void initialize(T* data, unsigned int size)
{
    for (unsigned int i{0}; i < size; ++i)
    {
        data[i] = static_cast<T>(i);
    }
}

template <class T>
bool compare(T const* data, T const* ref, unsigned int size)
{
    for (unsigned int i{0}; i < size; ++i)
    {
        if (data[i] != ref[i])
        {
            std::cout << i << " " << data[i] << " " << ref[i] << std::endl;
            return false;
        }
    }

    return true;
}

template <class T>
void print(T const* data, T const* ref, unsigned int size)
{
    for (unsigned int i{0}; i < size; ++i)
    {
        std::cout << i << " " << data[i] << " " << ref[i] << std::endl;
    }
}

int main()
{
    // Create CUDA stream.
    hipStream_t stream;
    CHECK_CUDA_ERROR(hipStreamCreate(&stream));

    using Element = int;

    unsigned int const M{2048}; // Number of columns.
    unsigned int const N{1024}; // Number of rows.

    // unsigned int const M{2049}; // Number of columns.
    // unsigned int const N{1021}; // Number of rows.

    auto const tensor_shape{cute::make_shape(N, M)};
    auto const tensor_shape_transposed{cute::make_shape(M, N)};

    thrust::host_vector<Element> h_src(cute::size(tensor_shape));
    thrust::host_vector<Element> h_dst(cute::size(tensor_shape_transposed));
    thrust::host_vector<Element> h_dst_ref(cute::size(tensor_shape_transposed));

    initialize(h_src.data(), h_src.size());
    transpose(h_src.data(), h_dst_ref.data(), M, N);

    thrust::device_vector<Element> d_src{h_src};
    thrust::device_vector<Element> d_dst{h_dst};

    auto const global_memory_layout_src{cute::make_layout(
        tensor_shape, cute::GenRowMajor{})}; // (N, M) : (M, 1)
    auto const global_memory_layout_dst{cute::make_layout(
        tensor_shape_transposed, cute::GenRowMajor{})}; // (M, N) : (N, 1)
    auto const global_memory_layout_dst_transposed{cute::make_layout(
        tensor_shape, cute::GenColMajor{})}; // (N, M) : (1, N)

    cute::print(global_memory_layout_src);
    std::cout << std::endl;
    cute::print(global_memory_layout_dst);
    std::cout << std::endl;
    cute::print(global_memory_layout_dst_transposed);
    std::cout << std::endl;

    auto const tensor_src{cute::make_tensor(
        cute::make_gmem_ptr(thrust::raw_pointer_cast(d_src.data())),
        global_memory_layout_src)};
    auto const tensor_dst{cute::make_tensor(
        cute::make_gmem_ptr(thrust::raw_pointer_cast(d_dst.data())),
        global_memory_layout_dst)};
    auto const tensor_dst_transposed{cute::make_tensor(
        cute::make_gmem_ptr(thrust::raw_pointer_cast(d_dst.data())),
        global_memory_layout_dst_transposed)};

    using TILE_SIZE_X = cute::Int<64>;
    using TILE_SIZE_Y = cute::Int<32>;

    constexpr auto block_shape{cute::make_shape(TILE_SIZE_Y{}, TILE_SIZE_X{})};
    constexpr auto block_shape_transposed{
        cute::make_shape(TILE_SIZE_X{}, TILE_SIZE_Y{})};

    auto const tiled_tensor_src{cute::tiled_divide(
        tensor_src, block_shape)}; // ((TILE_SIZE_Y, TILE_SIZE_X), N /
                                   // TILE_SIZE_Y, M / TILE_SIZE_X)
    auto const tiled_tensor_dst_transposed{cute::tiled_divide(
        tensor_dst_transposed, block_shape)}; // ((TILE_SIZE_Y, TILE_SIZE_X), N
                                              // / TILE_SIZE_Y, M / TILE_SIZE_X)
    cute::print(tiled_tensor_src);
    std::cout << std::endl;
    cute::print(tiled_tensor_dst_transposed);
    std::cout << std::endl;

    auto const g_src_example{
        tiled_tensor_src(cute::make_coord(cute::_, cute::_), 0, 0)};
    auto const g_dst_example{
        tiled_tensor_dst_transposed(cute::make_coord(cute::_, cute::_), 0, 0)};
    std::cout << "----------------" << std::endl;
    cute::print(g_src_example);
    std::cout << std::endl;
    cute::print(g_dst_example);
    std::cout << std::endl;
    // std::cout << "Make identity tensor" << std::endl;
    // auto const make_identity_tensor(g_src_example);

    using THREAD_BLOCK_SIZE_X = cute::Int<32>;
    using THREAD_BLOCK_SIZE_Y = cute::Int<8>;

    CUTE_STATIC_ASSERT(TILE_SIZE_X::value % THREAD_BLOCK_SIZE_X::value == 0,
                  "TILE_SIZE_X must be divisible by THREAD_BLOCK_SIZE_X");
    CUTE_STATIC_ASSERT(TILE_SIZE_Y::value % THREAD_BLOCK_SIZE_Y::value == 0,
                  "TILE_SIZE_Y must be divisible by THREAD_BLOCK_SIZE_Y");

    constexpr auto thread_block_shape{
        cute::make_shape(THREAD_BLOCK_SIZE_Y{}, THREAD_BLOCK_SIZE_X{})};
    constexpr auto thread_layout{
        cute::make_layout(thread_block_shape, cute::GenRowMajor{})};

    auto const thread_tile_src_example{
        cute::local_partition(g_src_example, thread_layout, 0)};
    auto const thread_tile_dst_transposed_example{
        cute::local_partition(g_dst_example, thread_layout, 0)};
    cute::print(thread_tile_src_example);
    std::cout << std::endl;
    cute::print(thread_tile_dst_transposed_example);
    std::cout << std::endl;

    auto const fragment_src_example{
        cute::make_tensor_like(thread_tile_src_example)};
    cute::print(fragment_src_example);
    std::cout << std::endl;

    dim3 const grid_dim{cute::size<2>(tiled_tensor_src),
                        cute::size<1>(tiled_tensor_src)};
    dim3 const thread_dim{cute::size(thread_layout)};

    transpose_naive<<<grid_dim, thread_dim, 0, stream>>>(
        tiled_tensor_src, tiled_tensor_dst_transposed, thread_layout);
    CHECK_LAST_CUDA_ERROR();

    CHECK_CUDA_ERROR(hipStreamSynchronize(stream));

    h_dst = d_dst;

    if (compare(h_dst.data(), h_dst_ref.data(), h_dst.size()))
    {
        std::cout << "Success!" << std::endl;
    }
    else
    {
        std::cout << "Failure!" << std::endl;
    }

    // print(h_dst.data(), h_dst_ref.data(), h_dst.size());
}