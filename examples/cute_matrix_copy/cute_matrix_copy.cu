#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cute/tensor.hpp>

#include "cute_matrix_copy.hpp"

template <class TENSOR_SRC, class TENSOR_DST, class THREAD_LAYOUT>
__global__ void copy(TENSOR_SRC tensor_src, TENSOR_DST tensor_dst_transposed, THREAD_LAYOUT)
{
    using Element = typename TENSOR_SRC::value_type;

    auto global_tile_src{tensor_src(cute::make_coord(cute::_, cute::_),
                                    blockIdx.y,
                                    blockIdx.x)}; // (TILE_SIZE_Y, TILE_SIZE_X)
    auto global_tile_dst_transposed{
        tensor_dst_transposed(cute::make_coord(cute::_, cute::_), blockIdx.y,
                              blockIdx.x)}; // (TILE_SIZE_Y, TILE_SIZE_X)

    auto thread_global_tile_src{cute::local_partition(
        global_tile_src, THREAD_LAYOUT{},
        threadIdx.x)}; // (THREAD_VALUE_SIZE_Y, THREAD_VALUE_SIZE_X)
    auto thread_global_tile_dst_transposed{cute::local_partition(
        global_tile_dst_transposed, THREAD_LAYOUT{},
        threadIdx.x)}; // (THREAD_VALUE_SIZE_Y, THREAD_VALUE_SIZE_X)
}