#include "hip/hip_runtime.h"
#include <iomanip>
#include <iostream>

#include <hip/hip_runtime.h>

#include <cute/algorithm/gemm.hpp>
#include <cute/tensor.hpp>

#include "cute_general_matrix_multiplication.hpp"

// Modified from the official CuTe example:
// https://github.com/NVIDIA/cutlass/blob/e1cd8c7866dd6de02b66a89879795e7d7301aacc/examples/cute/tutorial/sgemm_1.cu#L52
// This implementation uses shared memory.
// This implementation can have shared memory bank conflicts when copying data
// from global memory to shared memory. This implementation does not register to
// cache the data from shared memory for local mma. This implementation does not
// explicitly use vectorized memory access. This implementation does not use
// TensorCore for mma.
template <class ProblemShape, class CtaTiler, class TA, class AStride,
          class ASmemLayout, class AThreadLayout, class TB, class BStride,
          class BSmemLayout, class BThreadLayout, class TC, class CStride,
          class CSmemLayout, class CThreadLayout, class Alpha, class Beta>
static __global__ void general_matrix_multiplication_naive(
    ProblemShape shape_MNK, CtaTiler cta_tiler, TA const* A, AStride stride_A,
    ASmemLayout smem_layout_A, AThreadLayout thread_layout_A, TB const* B,
    BStride stride_B, BSmemLayout smem_layout_B, BThreadLayout thread_layout_B,
    TC* C, CStride stride_C, CSmemLayout, CThreadLayout thread_layout_C,
    Alpha alpha, Beta beta)
{
    CUTE_STATIC_ASSERT_V(cute::rank(shape_MNK) == cute::Int<3>{}); // (M, N, K)
    CUTE_STATIC_ASSERT_V(cute::rank(cta_tiler) ==
                         cute::Int<3>{}); // (BLK_M, BLK_N, BLK_K)

    // Thread layouts have to be static.
    CUTE_STATIC_ASSERT_V(cute::is_static<AThreadLayout>{});
    CUTE_STATIC_ASSERT_V(cute::is_static<BThreadLayout>{});
    CUTE_STATIC_ASSERT_V(cute::is_static<CThreadLayout>{});

    // Different thread layouts have to have the same number of threads.
    CUTE_STATIC_ASSERT_V(cute::size(thread_layout_A) ==
                         cute::size(thread_layout_B));
    CUTE_STATIC_ASSERT_V(cute::size(thread_layout_A) ==
                         cute::size(thread_layout_C));

    // CTA tiler has to be static.
    CUTE_STATIC_ASSERT_V(cute::is_static<CtaTiler>{});

    // CTA tiler has to be divisible by the thread layouts.
    CUTE_STATIC_ASSERT_V(cute::size<0>(cta_tiler) %
                             cute::size<0>(thread_layout_A) ==
                         cute::Int<0>{}); // BLK_M % THR_M == 0
    CUTE_STATIC_ASSERT_V(cute::size<2>(cta_tiler) %
                             cute::size<1>(thread_layout_A) ==
                         cute::Int<0>{}); // BLK_K % THR_K == 0
    CUTE_STATIC_ASSERT_V(cute::size<1>(cta_tiler) %
                             cute::size<0>(thread_layout_B) ==
                         cute::Int<0>{}); // BLK_N % THR_N == 0
    CUTE_STATIC_ASSERT_V(cute::size<2>(cta_tiler) %
                             cute::size<1>(thread_layout_B) ==
                         cute::Int<0>{}); // BLK_K % THR_K == 0
    CUTE_STATIC_ASSERT_V(cute::size<0>(cta_tiler) %
                             cute::size<0>(thread_layout_C) ==
                         cute::Int<0>{}); // BLK_M % THR_M == 0
    CUTE_STATIC_ASSERT_V(cute::size<1>(cta_tiler) %
                             cute::size<1>(thread_layout_C) ==
                         cute::Int<0>{}); // BLK_N % THR_N == 0

    // Shared memory layouts have to be static.
    CUTE_STATIC_ASSERT_V(cute::is_static<ASmemLayout>{});
    CUTE_STATIC_ASSERT_V(cute::is_static<BSmemLayout>{});
    CUTE_STATIC_ASSERT_V(cute::is_static<CSmemLayout>{});

    // Shared memory layouts have to match CTA tiler.
    CUTE_STATIC_ASSERT_V(cute::size<0>(smem_layout_A) ==
                         cute::size<0>(cta_tiler)); // BLK_M
    CUTE_STATIC_ASSERT_V(cute::size<1>(smem_layout_A) ==
                         cute::size<2>(cta_tiler)); // BLK_K
    CUTE_STATIC_ASSERT_V(cute::size<0>(smem_layout_B) ==
                         cute::size<1>(cta_tiler)); // BLK_N
    CUTE_STATIC_ASSERT_V(cute::size<1>(smem_layout_B) ==
                         cute::size<2>(cta_tiler)); // BLK_K

    // Shared memory layouts have to be divisible by the thread layouts.
    CUTE_STATIC_ASSERT_V(cute::size<0>(smem_layout_A) %
                             cute::size<0>(thread_layout_A) ==
                         cute::Int<0>{}); // BLK_M % THR_M == 0
    CUTE_STATIC_ASSERT_V(cute::size<1>(smem_layout_A) %
                             cute::size<1>(thread_layout_A) ==
                         cute::Int<0>{}); // BLK_K % THR_K == 0
    CUTE_STATIC_ASSERT_V(cute::size<0>(smem_layout_B) %
                             cute::size<0>(thread_layout_B) ==
                         cute::Int<0>{}); // BLK_N % THR_N == 0
    CUTE_STATIC_ASSERT_V(cute::size<1>(smem_layout_B) %
                             cute::size<1>(thread_layout_B) ==
                         cute::Int<0>{}); // BLK_K % THR_K == 0

    // Full tensor.
    // There are four scenarios for the full tensor.
    // 1. The shape of A is (M, K) and the shape of B is (K, N).
    //    Then A is (M, K) column-major and B is (K, N) column-major.
    //    Then A is (M, K) column-major and B is (N, K) row-major.
    // 2. The shape of transposed A is (M, K) and the shape of B is (K, N).
    //    Then A is (K, M) column-major and B is (K, N) column-major.
    //    Then A is (M, K) row-major and B is (N, K) row-major.
    // 3. The shape of A is (M, K) and the shape of transposed B is (K, N).
    //    Then A is (M, K) column-major and B is (N, K) column-major.
    // 4. The shape of transposed A is (M, K) and the shape of transposed B is
    // (K, N).
    //    Then A is (K, M) column-major and B is (N, K) column-major.
    //    Then A is (M, K) row-major and B is (N, K) column-major.
    auto global_full_tensor_A{cute::make_tensor(cute::make_gmem_ptr(A),
                                                cute::select<0, 2>(shape_MNK),
                                                stride_A)}; // (M, K)
    auto global_full_tensor_B{cute::make_tensor(cute::make_gmem_ptr(B),
                                                cute::select<1, 2>(shape_MNK),
                                                stride_B)}; // (N, K)
    // C is always (M, N) column-major.
    auto global_full_tensor_C{cute::make_tensor(cute::make_gmem_ptr(C),
                                                cute::select<0, 1>(shape_MNK),
                                                stride_C)}; // (M, N)

    // CTA index.
    // We used 3D index instead of 2D index because, as we will see later,
    // it will be convenient for the block selection, especially for the input
    // tensors A and B.
    auto cta_coord{
        cute::make_coord(blockIdx.x, blockIdx.y, cute::_)}; // (m, n, :)

    // Block selection.
    // With Step<_1, X, _1>{}, the second mode in the cta_tiler is ignored,
    // thus the tiler becomes (BLK_M, BLK_K).
    // In addition, because the the second mode is ignored, the cta_coord
    // becomes (m, :). So we will not select in the second mode.
    // The resulting local_tile is (BLK_M, BLK_K, k) where k is the number of
    // tiles to repeat and BLK_K * k = K if K is divisible by BLK_K.
    auto global_block_tensor_A{
        cute::local_tile(global_full_tensor_A, cta_tiler, cta_coord,
                         cute::Step<cute::Int<1>, cute::X,
                                    cute::Int<1>>{})}; // (BLK_M, BLK_K, k)
    // With Step<X, _1, _1>{}, the first mode in the cta_tiler is ignored,
    // thus the tiler becomes (BLK_N, BLK_K).
    // In addition, because the the first mode is ignored, the cta_coord
    // becomes (n, :). So we will not select in the first mode.
    // The resulting local_tile is (BLK_N, BLK_K, k) where k is the number of
    // tiles to repeat and BLK_K * k = K if K is divisible by BLK_K.
    auto global_block_tensor_B{
        cute::local_tile(global_full_tensor_B, cta_tiler, cta_coord,
                         cute::Step<cute::X, cute::Int<1>,
                                    cute::Int<1>>{})}; // (BLK_N, BLK_K, k)
    // With Step<_1, _1, X>{}, the third mode in the cta_tiler is ignored,
    // thus the tiler becomes (BLK_M, BLK_N).
    // In addition, because the the third mode is ignored, the cta_coord
    // becomes (m, n). So we will not select in the third mode.
    // The resulting local_tile is (BLK_M, BLK_N).
    auto global_block_tensor_C{cute::local_tile(
        global_full_tensor_C, cta_tiler, cta_coord,
        cute::Step<cute::Int<1>, cute::Int<1>, cute::X>{})}; // (BLK_M, BLK_N)

    // Shared memory buffers.
    __shared__ TA smem_A[cute::cosize_v<ASmemLayout>];
    __shared__ TB smem_B[cute::cosize_v<BSmemLayout>];
    // smem_layout_A and smem_layout_B are always column-major.
    // TODO: Add CUTE_STATIC_ASSERT to ensure the above conditions.
    auto smem_tensor_A{cute::make_tensor(cute::make_smem_ptr(smem_A),
                                         smem_layout_A)}; // (BLK_M, BLK_K)
    auto smem_tensor_B{cute::make_tensor(cute::make_smem_ptr(smem_B),
                                         smem_layout_B)}; // (BLK_N, BLK_K)

    // Partition the global_block_tensor_A and global_block_tensor_B across the
    // threads using the thread layout thread_layout_A and thread_layout_B.
    // Partition the smem_tensor_A and smem_tensor_B across the threads. This
    // will be used for copying the data from global memory to shared memory for
    // data reuse. Inner partition. This can ensure the memory access is
    // coalesced.
    auto thread_layout_A_global_block_tensor_A{cute::local_partition(
        global_block_tensor_A, thread_layout_A,
        threadIdx.x)}; // (BLK_M / THR_M, BLK_K / THR_K, k)
    auto thread_layout_B_global_block_tensor_B{cute::local_partition(
        global_block_tensor_B, thread_layout_B,
        threadIdx.x)}; // (BLK_N / THR_N, BLK_K / THR_K, k)
    auto thread_layout_A_smem_tensor_A{
        cute::local_partition(smem_tensor_A, thread_layout_A,
                              threadIdx.x)}; // (BLK_M / THR_M, BLK_K / THR_K)
    auto thread_layout_B_smem_tensor_B{
        cute::local_partition(smem_tensor_B, thread_layout_B,
                              threadIdx.x)}; // (BLK_N / THR_N, BLK_K / THR_K)

    CUTE_STATIC_ASSERT_V(
        cute::size<0>(thread_layout_A_global_block_tensor_A) ==
        cute::size<0>(thread_layout_A_smem_tensor_A)); // BLK_M / THR_M
    CUTE_STATIC_ASSERT_V(
        cute::size<1>(thread_layout_A_global_block_tensor_A) ==
        cute::size<1>(thread_layout_A_smem_tensor_A)); // BLK_K / THR_K
    CUTE_STATIC_ASSERT_V(
        cute::size<0>(thread_layout_B_global_block_tensor_B) ==
        cute::size<0>(thread_layout_B_smem_tensor_B)); // BLK_N / THR_N
    CUTE_STATIC_ASSERT_V(
        cute::size<1>(thread_layout_B_global_block_tensor_B) ==
        cute::size<1>(thread_layout_B_smem_tensor_B)); // BLK_K / THR_K

    // Partition the smem_tensor_A and smem_tensor_B across the threads using
    // the thread layout thread_layout_C. Partition the global_block_tensor_C
    // across the threads. This will be used for the gemm computation. Inner
    // partition. Partition the smem_tensor_A (BLK_M, BLK_K) by the rows of
    // thread_layout_C. Different threads in the same column of thread_layout_C
    // will read the same data from smem_tensor_A. With Step<_1, X>{}, the
    // second mode in the thread_layout_C layout is ignored.
    // The threads in the same warp will read contiguous data from smem_tensor_A
    // resulting in free of shared memory bank conflict.
    auto thread_layout_C_smem_tensor_A{cute::local_partition(
        smem_tensor_A, thread_layout_C, threadIdx.x,
        cute::Step<cute::Int<1>, cute::X>{})}; // (BLK_M / THR_M,
                                               // BLK_K)
    // Partition the smem_tensor_B (BLK_N, BLK_K) by the cols of
    // thread_layout_C. Different threads in the same row of thread_layout_C
    // will read the same data from smem_tensor_B. With Step<X, _1>{}, the first
    // mode in the thread_layout_C layout is ignored.
    // The threads in the same warp will read the same data from the same
    // location on smem_tensor_B resulting in a broadcast and no efficiency
    // loss.
    auto thread_layout_C_smem_tensor_B{cute::local_partition(
        smem_tensor_B, thread_layout_C, threadIdx.x,
        cute::Step<cute::X, cute::Int<1>>{})}; // (BLK_N / THR_N,
                                               // BLK_K)
    // Partition the global_block_tensor_C (BLK_M, BLK_N) by the tile of
    // thread_layout_C.
    auto thread_layout_C_global_block_tensor_C{cute::local_partition(
        global_block_tensor_C, thread_layout_C, threadIdx.x,
        cute::Step<cute::Int<1>, cute::Int<1>>{})}; // (BLK_M / THR_M, BLK_N /
                                                    // THR_N)
    // This is the same as the above.
    // auto thread_layout_C_global_block_tensor_C{
    //     cute::local_partition(global_block_tensor_C, thread_layout_C,
    //                           threadIdx.x)}; // (BLK_M / THR_M, BLK_N /
    //                           THR_N)

    // Allocate the accumulators.
    // The layout is automatically compacted to the smallest possible layout to
    // avoid unnecessary memory/register usage.
    auto thread_layout_C_register_tensor_C{cute::make_tensor_like(
        thread_layout_C_global_block_tensor_C)}; // (BLK_M / THR_M, BLK_N /
                                                 // THR_N)

    CUTE_STATIC_ASSERT_V(
        cute::size<0>(thread_layout_C_smem_tensor_A) ==
        cute::size<0>(thread_layout_C_register_tensor_C)); // BLK_M / THR_M
    CUTE_STATIC_ASSERT_V(
        cute::size<0>(thread_layout_C_smem_tensor_B) ==
        cute::size<1>(thread_layout_C_register_tensor_C)); // BLK_N / THR_N
    CUTE_STATIC_ASSERT_V(
        cute::size<0>(thread_layout_C_global_block_tensor_C) ==
        cute::size<0>(thread_layout_C_register_tensor_C)); // BLK_M / THR_M
    CUTE_STATIC_ASSERT_V(
        cute::size<1>(thread_layout_C_global_block_tensor_C) ==
        cute::size<1>(thread_layout_C_register_tensor_C)); // BLK_N / THR_N

    // Clear the accumulators.
    cute::clear(thread_layout_C_register_tensor_C);

#ifdef NO_BOUNDS_CHECK

#else
    // Create predicate tensors.
    auto thread_layout_A_predicate_tensor_A{cute::make_tensor<bool>(
        cute::make_shape(cute::size<0>(thread_layout_A_global_block_tensor_A),
                         cute::size<1>(thread_layout_A_global_block_tensor_A)),
        cute::make_stride(cute::Int<1>{}, cute::Int<0>{}))};
    auto thread_layout_B_predicate_tensor_B{cute::make_tensor<bool>(
        cute::make_shape(cute::size<0>(thread_layout_B_global_block_tensor_B),
                         cute::size<1>(thread_layout_B_global_block_tensor_B)),
        cute::make_stride(cute::Int<1>{}, cute::Int<0>{}))};
    auto thread_layout_C_predicate_tensor_C{cute::make_tensor<bool>(
        cute::make_shape(cute::size<0>(thread_layout_C_global_block_tensor_C),
                         cute::size<1>(thread_layout_C_global_block_tensor_C)),
        cute::make_stride(
            cute::Int<1>{},
            cute::size<0>(thread_layout_C_global_block_tensor_C)))};
    // Create identity tensors.
    auto identity_tensor_A{cute::make_identity_tensor(cute::make_shape(
        cute::size<0>(smem_tensor_A), cute::size<1>(smem_tensor_A)))};
    auto identity_tensor_B{cute::make_identity_tensor(cute::make_shape(
        cute::size<0>(smem_tensor_B), cute::size<1>(smem_tensor_B)))};
    auto identity_tensor_C{cute::make_identity_tensor(
        cute::make_shape(cute::size<0>(global_block_tensor_C),
                         cute::size<1>(global_block_tensor_C)))};
    auto thread_layout_A_identity_tensor_A{
        cute::local_partition(identity_tensor_A, thread_layout_A,
                              threadIdx.x)}; // (BLK_M / THR_M, BLK_K / THR_K)
    auto thread_layout_B_identity_tensor_B{
        cute::local_partition(identity_tensor_B, thread_layout_B,
                              threadIdx.x)}; // (BLK_N / THR_N, BLK_K / THR_K)
    auto thread_layout_C_identity_tensor_C{
        cute::local_partition(identity_tensor_C, thread_layout_C,
                              threadIdx.x)}; // (BLK_M / THR_M, BLK_N / THR_N)

    CUTE_UNROLL
    for (auto m{0}; m < cute::size<0>(thread_layout_A_predicate_tensor_A); ++m)
    {
        thread_layout_A_predicate_tensor_A(m, 0) =
            cute::get<0>(thread_layout_A_identity_tensor_A(m, 0)) +
                blockIdx.x * cute::size<0>(smem_tensor_A) <
            cute::size<0>(shape_MNK);
    }
    CUTE_UNROLL
    for (auto n{0}; n < cute::size<0>(thread_layout_B_predicate_tensor_B); ++n)
    {
        thread_layout_B_predicate_tensor_B(n, 0) =
            cute::get<0>(thread_layout_B_identity_tensor_B(n, 0)) +
                blockIdx.y * cute::size<0>(smem_tensor_B) <
            cute::size<1>(shape_MNK);
    }
    CUTE_UNROLL
    for (auto m{0}; m < cute::size<0>(thread_layout_C_predicate_tensor_C); ++m)
    {
        CUTE_UNROLL
        for (auto n{0}; n < cute::size<1>(thread_layout_C_predicate_tensor_C);
             ++n)
        {
            thread_layout_C_predicate_tensor_C(m, n) =
                cute::get<0>(thread_layout_C_identity_tensor_C(m, n)) +
                        blockIdx.x * cute::size<0>(global_block_tensor_C) <
                    cute::size<0>(shape_MNK) &&
                cute::get<1>(thread_layout_C_identity_tensor_C(m, n)) +
                        blockIdx.y * cute::size<1>(global_block_tensor_C) <
                    cute::size<1>(shape_MNK);
        }
    }
#endif

    // Perform the gemm computation loop.
    auto const num_tiles_k{cute::size<2>(global_block_tensor_A)}; // k

    for (auto tile_idx_k{0}; tile_idx_k < num_tiles_k; ++tile_idx_k)
    {
#ifdef NO_BOUNDS_CHECK
        cute::copy(
            thread_layout_A_global_block_tensor_A(cute::_, cute::_, tile_idx_k),
            thread_layout_A_smem_tensor_A);
        cute::copy(
            thread_layout_B_global_block_tensor_B(cute::_, cute::_, tile_idx_k),
            thread_layout_B_smem_tensor_B);
#else
        // Clear the shared memory buffers.
        // This is necessary when predicates are used for copying data from
        // global memory to shared memory so that mma will not be affected by
        // the previous data in the unwanted region.
        cute::clear(thread_layout_A_smem_tensor_A);
        cute::clear(thread_layout_B_smem_tensor_B);

        CUTE_UNROLL
        for (auto k{0};
             k < cute::size<1>(thread_layout_A_global_block_tensor_A); ++k)
        {
            // Check the K dimension.
            if (cute::get<1>(thread_layout_A_identity_tensor_A(0, k)) +
                    tile_idx_k * cute::size<1>(smem_tensor_A) <
                cute::size<2>(shape_MNK))
            {
                cute::copy_if(thread_layout_A_predicate_tensor_A,
                              thread_layout_A_global_block_tensor_A(cute::_, k,
                                                                    tile_idx_k),
                              thread_layout_A_smem_tensor_A(cute::_, k));
            }
        }
        CUTE_UNROLL
        for (auto k{0};
             k < cute::size<1>(thread_layout_B_global_block_tensor_B); ++k)
        {
            // Check the K dimension.
            if (cute::get<1>(thread_layout_B_identity_tensor_B(0, k)) +
                    tile_idx_k * cute::size<1>(smem_tensor_B) <
                cute::size<2>(shape_MNK))
            {
                cute::copy_if(thread_layout_B_predicate_tensor_B,
                              thread_layout_B_global_block_tensor_B(cute::_, k,
                                                                    tile_idx_k),
                              thread_layout_B_smem_tensor_B(cute::_, k));
            }
        }
#endif
        // // Copy the data from global memory to shared memory for data reuse.
        // // This is the only place where shared memory bank conflicts can
        // happen,
        // // depending on the tensor layouts on the global memory.
        // // Copy the data from global_block_tensor_A to smem_tensor_A.
        // cute::copy(
        //     thread_layout_A_global_block_tensor_A(cute::_, cute::_,
        //     tile_idx_k), thread_layout_A_smem_tensor_A); // (BLK_M / THR_M,
        //     BLK_K / THR_K) ->
        //                                     // (BLK_M / THR_M, BLK_K / THR_K)
        // // Copy the data from global_block_tensor_B to smem_tensor_B.
        // cute::copy(
        //     thread_layout_B_global_block_tensor_B(cute::_, cute::_,
        //     tile_idx_k), thread_layout_B_smem_tensor_B); // (BLK_N / THR_N,
        //     BLK_K / THR_K) ->
        //                                     // (BLK_N / THR_N, BLK_K / THR_K)

        // Synchronize the threads to ensure the data copy is completed.
        cute::cp_async_fence();
        cute::cp_async_wait<0>();
        __syncthreads();

        // Compute gemm on thread_layout_C thread-partitioned smem.
        // This implicitly uses the UniversalFMA GEMM atom.
        cute::gemm(thread_layout_C_smem_tensor_A, thread_layout_C_smem_tensor_B,
                   thread_layout_C_register_tensor_C); // (BLK_M / THR_M, BLK_N
                                                       // / THR_N) += (BLK_M /
                                                       // THR_M, BLK_K) * (BLK_N
                                                       // / THR_N, BLK_K)
        // This is equivalent to the above.
        // auto mma_atom{cute::MMA_Atom<cute::UniversalFMA<TC, TA, TB, TC>>{}};
        // cute::gemm(mma_atom, thread_layout_C_smem_tensor_A,
        //            thread_layout_C_smem_tensor_B,
        //            thread_layout_C_register_tensor_C); // (BLK_M / THR_M,
        //            BLK_N
        //                                                // / THR_N) += (BLK_M
        //                                                /
        //                                                // THR_M, BLK_K) *
        //                                                (BLK_N
        //                                                // / THR_N, BLK_K)

        __syncthreads();
    }

    // Scale and accumulate the result from the register tensor to the global
    // block tensor.
    // cute::axpby(
    //     alpha, thread_layout_C_register_tensor_C, beta,
    //     thread_layout_C_global_block_tensor_C); // (BLK_M / THR_M, BLK_N /
    //                                             // THR_N) = alpha * (BLK_M /
    //                                             // THR_M, BLK_N / THR_N) +
    //                                             beta
    //                                             // * (BLK_M / THR_M, BLK_N /
    //                                             // THR_N)
#ifdef NO_BOUNDS_CHECK
    cute::axpby(alpha, thread_layout_C_register_tensor_C, beta,
                thread_layout_C_global_block_tensor_C);
#else
    cute::axpby(alpha, thread_layout_C_register_tensor_C, beta,
                thread_layout_C_global_block_tensor_C,
                thread_layout_C_predicate_tensor_C);
#endif
}

template <class TA, class TB, class TC, class Alpha, class Beta, class AStride,
          class BStride, class CStride>
static hipError_t gemm_base(int m, int n, int k, Alpha alpha, TA const* A,
                             int ldA, TB const* B, int ldB, Beta beta, TC* C,
                             int ldC, AStride stride_A, BStride stride_B,
                             CStride stride_C, hipStream_t stream)
{
    // Define GEMM shape.
    auto const M{m};
    auto const N{n};
    auto const K{k};
    auto const gemm_shape{cute::make_shape(M, N, K)}; // (M, N, K)

    // Define CTA size.
    auto const bM{cute::Int<128 * 2 / sizeof(TA)>{}};
    auto const bN{cute::Int<128 * 2 / sizeof(TB)>{}};
    auto const bK{cute::Int<32>{}};
    auto const cta_tiler{cute::make_shape(bM, bN, bK)}; // (BLK_M, BLK_N, BLK_K)

    // Define smem layouts.
    // smem_layout_A is (BLK_M, BLK_K) column-major.
    // smem_layout_B is (BLK_N, BLK_K) column-major.
    // smem_layout_C is (BLK_M, BLK_N) column-major.
    auto const smem_shape_A{cute::make_shape(bM, bK)}; // (BLK_M, BLK_K)
    auto const smem_stride_A{cute::make_stride(
        cute::Int<1>{}, cute::size<0>(smem_shape_A))}; // column-major
    auto const smem_layout_A{
        cute::make_layout(smem_shape_A, smem_stride_A)}; // (BLK_M, BLK_K)
    auto const smem_shape_B{cute::make_shape(bN, bK)};   // (BLK_N, BLK_K)
    auto const smem_stride_B{cute::make_stride(
        cute::Int<1>{}, cute::size<0>(smem_shape_B))}; // column-major
    auto const smem_layout_B{
        cute::make_layout(smem_shape_B, smem_stride_B)}; // (BLK_N, BLK_K)
    auto const smem_shape_C{cute::make_shape(bM, bN)};   // (BLK_M, BLK_N)
    auto const smem_stride_C{cute::make_stride(
        cute::Int<1>{}, cute::size<0>(smem_shape_C))}; // column-major
    auto const smem_layout_C{
        cute::make_layout(smem_shape_C, smem_stride_C)}; // (BLK_M, BLK_N)

    // Define thread layouts.
    auto const thread_shape_A{
        cute::make_shape(cute::Int<16>{}, cute::Int<8>{})}; // (THR_M, THR_K)
    auto const thread_shape_B{
        cute::make_shape(cute::Int<16>{}, cute::Int<8>{})}; // (THR_N, THR_K)
    auto const thread_shape_C{
        cute::make_shape(cute::Int<16>{}, cute::Int<8>{})}; // (THR_M, THR_N)
    auto const thread_stride_A{cute::make_stride(
        cute::Int<1>{}, cute::size<0>(thread_shape_A))}; // column-major
    auto const thread_stride_B{cute::make_stride(
        cute::Int<1>{}, cute::size<0>(thread_shape_B))}; // column-major
    auto const thread_stride_C{cute::make_stride(
        cute::Int<1>{}, cute::size<0>(thread_shape_C))}; // column-major
    auto const thread_layout_A{
        cute::make_layout(thread_shape_A, thread_stride_A)}; // (THR_M, THR_K)
    auto const thread_layout_B{
        cute::make_layout(thread_shape_B, thread_stride_B)}; // (THR_N, THR_K)
    auto const thread_layout_C{
        cute::make_layout(thread_shape_C, thread_stride_C)}; // (THR_M, THR_N)
    CUTE_STATIC_ASSERT_V(cute::size(thread_layout_A) ==
                         cute::size(thread_layout_B));
    CUTE_STATIC_ASSERT_V(cute::size(thread_layout_A) ==
                         cute::size(thread_layout_C));

    // Launch the kernel.
    dim3 const block_dims{
        static_cast<unsigned int>(cute::size(thread_layout_C))};
    dim3 const grid_dims{
        static_cast<unsigned int>(cute::size(cute::ceil_div(M, bM))),
        static_cast<unsigned int>(cute::size(cute::ceil_div(N, bN)))};
    general_matrix_multiplication_naive<<<grid_dims, block_dims, 0, stream>>>(
        gemm_shape, cta_tiler, A, stride_A, smem_layout_A, thread_layout_A, B,
        stride_B, smem_layout_B, thread_layout_B, C, stride_C, smem_layout_C,
        thread_layout_C, alpha, beta);

    return hipGetLastError();
}

// The shape of A is (M, K) and the shape of B is (K, N).
// Then A is (M, K) column-major and B is (K, N) column-major.
// Then A is (M, K) column-major and B is (N, K) row-major.
template <class TA, class TB, class TC, class Alpha, class Beta>
static hipError_t gemm_nn(int m, int n, int k, Alpha alpha, TA const* A,
                           int ldA, TB const* B, int ldB, Beta beta, TC* C,
                           int ldC, hipStream_t stream)
{
    // Define global memory layouts.
    // A is (M, K) column-major.
    auto const stride_A{cute::make_stride(cute::Int<1>{}, ldA)}; // column-major
    // B is (N, K) row-major.
    auto const stride_B{cute::make_stride(ldB, cute::Int<1>{})}; // row-major
    // C is (M, N) column-major.
    auto const stride_C{cute::make_stride(cute::Int<1>{}, ldC)}; // column-major

    return gemm_base(m, n, k, alpha, A, ldA, B, ldB, beta, C, ldC, stride_A,
                     stride_B, stride_C, stream);
}

// The shape of A is (M, K) and the shape of transposed B is (K, N).
// Then A is (M, K) column-major and B is (N, K) column-major.
// The smem_A is (BLK_M, BLK_K) column-major and smem_B is (BLK_N, BLK_K)
// column-major.
template <class TA, class TB, class TC, class Alpha, class Beta>
static hipError_t gemm_nt(int m, int n, int k, Alpha alpha, TA const* A,
                           int ldA, TB const* B, int ldB, Beta beta, TC* C,
                           int ldC, hipStream_t stream)
{
    // Define global memory layouts.
    // A is (M, K) column-major.
    auto const stride_A{cute::make_stride(cute::Int<1>{}, ldA)}; // column-major
    // B is (N, K) column-major.
    auto const stride_B{cute::make_stride(cute::Int<1>{}, ldB)}; // column-major
    // C is (M, N) column-major.
    auto const stride_C{cute::make_stride(cute::Int<1>{}, ldC)}; // column-major

    return gemm_base(m, n, k, alpha, A, ldA, B, ldB, beta, C, ldC, stride_A,
                     stride_B, stride_C, stream);
}

// The shape of transposed A is (M, K) and the shape of B is (K, N).
// Then A is (K, M) column-major and B is (K, N) column-major.
// Then A is (M, K) row-major and B is (N, K) row-major.
template <class TA, class TB, class TC, class Alpha, class Beta>
static hipError_t gemm_tn(int m, int n, int k, Alpha alpha, TA const* A,
                           int ldA, TB const* B, int ldB, Beta beta, TC* C,
                           int ldC, hipStream_t stream)
{
    // Define global memory layouts.
    // A is (M, K) row-major.
    auto const stride_A{cute::make_stride(ldA, cute::Int<1>{})}; // row-major
    // B is (N, K) row-major.
    auto const stride_B{cute::make_stride(ldB, cute::Int<1>{})}; // row-major
    // C is (M, N) column-major.
    auto const stride_C{cute::make_stride(cute::Int<1>{}, ldC)}; // column-major

    return gemm_base(m, n, k, alpha, A, ldA, B, ldB, beta, C, ldC, stride_A,
                     stride_B, stride_C, stream);
}

// The shape of transposed A is (M, K) and the shape of transposed B is (K, N).
//    Then A is (K, M) column-major and B is (N, K) column-major.
//    Then A is (M, K) row-major and B is (N, K) column-major.
template <class TA, class TB, class TC, class Alpha, class Beta>
static hipError_t gemm_tt(int m, int n, int k, Alpha alpha, TA const* A,
                           int ldA, TB const* B, int ldB, Beta beta, TC* C,
                           int ldC, hipStream_t stream)
{
    // Define global memory layouts.
    // A is (M, K) row-major.
    auto const stride_A{cute::make_stride(ldA, cute::Int<1>{})}; // row-major
    // B is (N, K) column-major.
    auto const stride_B{cute::make_stride(cute::Int<1>{}, ldB)}; // column-major
    // C is (M, N) column-major.
    auto const stride_C{cute::make_stride(cute::Int<1>{}, ldC)}; // column-major

    return gemm_base(m, n, k, alpha, A, ldA, B, ldB, beta, C, ldC, stride_A,
                     stride_B, stride_C, stream);
}

template <class TA, class TB, class TC, class Alpha, class Beta>
hipError_t launch_gemm_naive(char transA, char transB, int m, int n, int k,
                              Alpha alpha, TA const* A, int ldA, TB const* B,
                              int ldB, Beta beta, TC* C, int ldC,
                              hipStream_t stream)
{
    if (transA == 'N' && transB == 'T')
    {
        return gemm_nt(m, n, k, alpha, A, ldA, B, ldB, beta, C, ldC, stream);
    }
    else if (transA == 'N' && transB == 'N')
    {
        return gemm_nn(m, n, k, alpha, A, ldA, B, ldB, beta, C, ldC, stream);
    }
    else if (transA == 'T' && transB == 'N')
    {
        return gemm_tn(m, n, k, alpha, A, ldA, B, ldB, beta, C, ldC, stream);
    }
    else if (transA == 'T' && transB == 'T')
    {
        return gemm_tt(m, n, k, alpha, A, ldA, B, ldB, beta, C, ldC, stream);
    }
    else
    {
        return hipErrorNotSupported;
    }
}

// Explicit instantiation
template hipError_t launch_gemm_naive<float, float, float, float, float>(
    char transA, char transB, int m, int n, int k, float alpha, float const* A,
    int ldA, float const* B, int ldB, float beta, float* C, int ldC,
    hipStream_t stream);
template hipError_t launch_gemm_naive<double, double, double, double, double>(
    char transA, char transB, int m, int n, int k, double alpha,
    double const* A, int ldA, double const* B, int ldB, double beta, double* C,
    int ldC, hipStream_t stream);
template hipError_t
launch_gemm_naive<cute::half_t, cute::half_t, cute::half_t, float, float>(
    char transA, char transB, int m, int n, int k, float alpha,
    cute::half_t const* A, int ldA, cute::half_t const* B, int ldB, float beta,
    cute::half_t* C, int ldC, hipStream_t stream);
template hipError_t launch_gemm_naive<cute::half_t, cute::half_t, cute::half_t,
                                       cute::half_t, cute::half_t>(
    char transA, char transB, int m, int n, int k, cute::half_t alpha,
    cute::half_t const* A, int ldA, cute::half_t const* B, int ldB,
    cute::half_t beta, cute::half_t* C, int ldC, hipStream_t stream);