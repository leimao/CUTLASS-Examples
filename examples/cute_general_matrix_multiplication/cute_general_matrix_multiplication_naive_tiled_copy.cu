#include "hip/hip_runtime.h"
#include <iomanip>
#include <iostream>

#include <hip/hip_runtime.h>

#include <cute/algorithm/gemm.hpp>
#include <cute/tensor.hpp>

#include "cute_general_matrix_multiplication.hpp"

constexpr int constexpr_log2(int n)
{
    return ((n < 2) ? 0 : 1 + constexpr_log2(n / 2));
}

// Tiled copy can allow vectorized memory access and improve kernel performance.
template <class ProblemShape, class CtaTiler, class TA, class AStride,
          class ASmemLayout, class AThreadLayout, class TiledCopyA, class TB,
          class BStride, class BSmemLayout, class BThreadLayout,
          class TiledCopyB, class TC, class CStride, class CSmemLayout,
          class CThreadLayout, class Alpha, class Beta>
static __global__ void general_matrix_multiplication_naive_tiled_copy(
    ProblemShape shape_MNK, CtaTiler cta_tiler, TA const* A, AStride stride_A,
    ASmemLayout smem_layout_A, AThreadLayout, TiledCopyA copy_A, TB const* B,
    BStride stride_B, BSmemLayout smem_layout_B, BThreadLayout,
    TiledCopyB copy_B, TC* C, CStride stride_C, CSmemLayout,
    CThreadLayout thread_layout_C, Alpha alpha, Beta beta)
{
    CUTE_STATIC_ASSERT_V(cute::rank(shape_MNK) == cute::Int<3>{}); // (M, N, K)
    CUTE_STATIC_ASSERT_V(cute::rank(cta_tiler) ==
                         cute::Int<3>{}); // (BLK_M, BLK_N, BLK_K)

    // Thread layouts have to be static.
    CUTE_STATIC_ASSERT_V(cute::is_static<AThreadLayout>{});
    CUTE_STATIC_ASSERT_V(cute::is_static<BThreadLayout>{});
    CUTE_STATIC_ASSERT_V(cute::is_static<CThreadLayout>{});

    // CTA tiler has to be static.
    CUTE_STATIC_ASSERT_V(cute::is_static<CtaTiler>{});

    // Shared memory layouts have to be static.
    CUTE_STATIC_ASSERT_V(cute::is_static<ASmemLayout>{});
    CUTE_STATIC_ASSERT_V(cute::is_static<BSmemLayout>{});
    CUTE_STATIC_ASSERT_V(cute::is_static<CSmemLayout>{});

    // Shared memory layouts have to match CTA tiler.
    CUTE_STATIC_ASSERT_V(cute::size<0>(smem_layout_A) ==
                         cute::size<0>(cta_tiler)); // BLK_M
    CUTE_STATIC_ASSERT_V(cute::size<1>(smem_layout_A) ==
                         cute::size<2>(cta_tiler)); // BLK_K
    CUTE_STATIC_ASSERT_V(cute::size<0>(smem_layout_B) ==
                         cute::size<1>(cta_tiler)); // BLK_N
    CUTE_STATIC_ASSERT_V(cute::size<1>(smem_layout_B) ==
                         cute::size<2>(cta_tiler)); // BLK_K

    // Full tensor.
    // There are four scenarios for the full tensor.
    // 1. The shape of A is (M, K) and the shape of B is (K, N).
    //    Then A is (M, K) column-major and B is (K, N) column-major.
    //    Then A is (M, K) column-major and B is (N, K) row-major.
    // 2. The shape of transposed A is (M, K) and the shape of B is (K, N).
    //    Then A is (K, M) column-major and B is (K, N) column-major.
    //    Then A is (M, K) row-major and B is (N, K) row-major.
    // 3. The shape of A is (M, K) and the shape of transposed B is (K, N).
    //    Then A is (M, K) column-major and B is (N, K) column-major.
    // 4. The shape of transposed A is (M, K) and the shape of transposed B is
    // (K, N).
    //    Then A is (K, M) column-major and B is (N, K) column-major.
    //    Then A is (M, K) row-major and B is (N, K) column-major.
    auto global_full_tensor_A{cute::make_tensor(cute::make_gmem_ptr(A),
                                                cute::select<0, 2>(shape_MNK),
                                                stride_A)}; // (M, K)
    auto global_full_tensor_B{cute::make_tensor(cute::make_gmem_ptr(B),
                                                cute::select<1, 2>(shape_MNK),
                                                stride_B)}; // (N, K)
    // C is always (M, N) column-major.
    auto global_full_tensor_C{cute::make_tensor(cute::make_gmem_ptr(C),
                                                cute::select<0, 1>(shape_MNK),
                                                stride_C)}; // (M, N)

    // CTA index.
    // We used 3D index instead of 2D index because, as we will see later,
    // it will be convenient for the block selection, especially for the input
    // tensors A and B.
    auto cta_coord{
        cute::make_coord(blockIdx.x, blockIdx.y, cute::_)}; // (m, n, :)

    // Block selection.
    // With Step<_1, X, _1>{}, the second mode in the cta_tiler is ignored,
    // thus the tiler becomes (BLK_M, BLK_K).
    // In addition, because the the second mode is ignored, the cta_coord
    // becomes (m, :). So we will not select in the second mode.
    // The resulting local_tile is (BLK_M, BLK_K, k) where k is the number of
    // tiles to repeat and BLK_K * k = K if K is divisible by BLK_K.
    auto global_block_tensor_A{
        cute::local_tile(global_full_tensor_A, cta_tiler, cta_coord,
                         cute::Step<cute::Int<1>, cute::X,
                                    cute::Int<1>>{})}; // (BLK_M, BLK_K, k)
    // With Step<X, _1, _1>{}, the first mode in the cta_tiler is ignored,
    // thus the tiler becomes (BLK_N, BLK_K).
    // In addition, because the the first mode is ignored, the cta_coord
    // becomes (n, :). So we will not select in the first mode.
    // The resulting local_tile is (BLK_N, BLK_K, k) where k is the number of
    // tiles to repeat and BLK_K * k = K if K is divisible by BLK_K.
    auto global_block_tensor_B{
        cute::local_tile(global_full_tensor_B, cta_tiler, cta_coord,
                         cute::Step<cute::X, cute::Int<1>,
                                    cute::Int<1>>{})}; // (BLK_N, BLK_K, k)
    // With Step<_1, _1, X>{}, the third mode in the cta_tiler is ignored,
    // thus the tiler becomes (BLK_M, BLK_N).
    // In addition, because the the third mode is ignored, the cta_coord
    // becomes (m, n). So we will not select in the third mode.
    // The resulting local_tile is (BLK_M, BLK_N).
    auto global_block_tensor_C{cute::local_tile(
        global_full_tensor_C, cta_tiler, cta_coord,
        cute::Step<cute::Int<1>, cute::Int<1>, cute::X>{})}; // (BLK_M, BLK_N)

    // Shared memory buffers.
    __shared__ TA smem_A[cute::cosize_v<ASmemLayout>];
    __shared__ TB smem_B[cute::cosize_v<BSmemLayout>];
    // sA and sB are always column-major.
    // TODO: Add static_assert to ensure the above conditions.
    auto smem_tensor_A{cute::make_tensor(cute::make_smem_ptr(smem_A),
                                         smem_layout_A)}; // (BLK_M, BLK_K)
    auto smem_tensor_B{cute::make_tensor(cute::make_smem_ptr(smem_B),
                                         smem_layout_B)}; // (BLK_N, BLK_K)

    // Partition via tiled copy.
    auto thread_copy_A{copy_A.get_slice(threadIdx.x)};
    auto thread_layout_A_global_block_tensor_A{thread_copy_A.partition_S(
        global_block_tensor_A)}; // (CPY, CPY_M, CPY_K, k)
    auto thread_layout_A_smem_tensor_A{
        thread_copy_A.partition_D(smem_tensor_A)}; // (CPY, CPY_M, CPY_K)
    auto thread_copy_B{copy_B.get_slice(threadIdx.x)};
    auto thread_layout_B_global_block_tensor_B{thread_copy_B.partition_S(
        global_block_tensor_B)}; // (CPY, CPY_N, CPY_K, k)
    auto thread_layout_B_smem_tensor_B{
        thread_copy_B.partition_D(smem_tensor_B)}; // (CPY, CPY_N, CPY_K)

    // Partition the smem_tensor_A and smem_tensor_B across the threads using
    // the thread layout thread_layout_C. Partition the global_block_tensor_C
    // across the threads. This will be used for the gemm computation. Inner
    // partition. Partition the smem_tensor_A (BLK_M, BLK_K) by the rows of
    // thread_layout_C. Different threads in the same column of thread_layout_C
    // will read the same data from smem_tensor_A. With Step<_1, X>{}, the
    // second mode in the thread_layout_C layout is ignored.
    // The threads in the same warp will read contiguous data from smem_tensor_A
    // resulting in free of shared memory bank conflict.
    auto thread_layout_C_smem_tensor_A{cute::local_partition(
        smem_tensor_A, thread_layout_C, threadIdx.x,
        cute::Step<cute::Int<1>, cute::X>{})}; // (BLK_M / THR_M,
                                               // BLK_K)
    // Partition the smem_tensor_B (BLK_N, BLK_K) by the cols of
    // thread_layout_C. Different threads in the same row of thread_layout_C
    // will read the same data from smem_tensor_B. With Step<X, _1>{}, the first
    // mode in the thread_layout_C layout is ignored.
    // The threads in the same warp will read the same data from the same
    // location on smem_tensor_B resulting in a broadcast and no efficiency
    // loss.
    auto thread_layout_C_smem_tensor_B{cute::local_partition(
        smem_tensor_B, thread_layout_C, threadIdx.x,
        cute::Step<cute::X, cute::Int<1>>{})}; // (BLK_N / THR_N,
                                               // BLK_K)
    // Partition the global_block_tensor_C (BLK_M, BLK_N) by the tile of
    // thread_layout_C.
    auto thread_layout_C_global_block_tensor_C{cute::local_partition(
        global_block_tensor_C, thread_layout_C, threadIdx.x,
        cute::Step<cute::Int<1>, cute::Int<1>>{})}; // (BLK_M / THR_M, BLK_N /
                                                    // THR_N)
    // This is the same as the above.
    // auto thread_layout_C_global_block_tensor_C{
    //     cute::local_partition(global_block_tensor_C, thread_layout_C,
    //                           threadIdx.x)}; // (BLK_M / THR_M, BLK_N /
    //                           THR_N)

    // Allocate the accumulators.
    // The layout is automatically compacted to the smallest possible layout to
    // avoid unnecessary memory/register usage.
    auto thread_layout_C_register_tensor_C{cute::make_tensor_like(
        thread_layout_C_global_block_tensor_C)}; // (BLK_M / THR_M, BLK_N /
                                                 // THR_N)

    CUTE_STATIC_ASSERT_V(
        cute::size<0>(thread_layout_C_smem_tensor_A) ==
        cute::size<0>(thread_layout_C_register_tensor_C)); // BLK_M / THR_M
    CUTE_STATIC_ASSERT_V(
        cute::size<0>(thread_layout_C_smem_tensor_B) ==
        cute::size<1>(thread_layout_C_register_tensor_C)); // BLK_N / THR_N
    CUTE_STATIC_ASSERT_V(
        cute::size<0>(thread_layout_C_global_block_tensor_C) ==
        cute::size<0>(thread_layout_C_register_tensor_C)); // BLK_M / THR_M
    CUTE_STATIC_ASSERT_V(
        cute::size<1>(thread_layout_C_global_block_tensor_C) ==
        cute::size<1>(thread_layout_C_register_tensor_C)); // BLK_N / THR_N

    // Clear the accumulators.
    cute::clear(thread_layout_C_register_tensor_C);

    // Create predicate tensors.
    // To simplify the implementation a little bit, we used 2D predicate tensors
    // which can take a little bit more register space.
    auto thread_layout_A_predicate_tensor_A{cute::make_tensor<bool>(
        cute::make_shape(cute::size<1>(thread_layout_A_global_block_tensor_A),
                         cute::size<2>(thread_layout_A_global_block_tensor_A)),
        cute::make_stride(
            cute::Int<1>{},
            cute::size<1>(thread_layout_A_global_block_tensor_A)))};
    auto thread_layout_B_predicate_tensor_B{cute::make_tensor<bool>(
        cute::make_shape(cute::size<1>(thread_layout_B_global_block_tensor_B),
                         cute::size<2>(thread_layout_B_global_block_tensor_B)),
        cute::make_stride(
            cute::Int<1>{},
            cute::size<1>(thread_layout_B_global_block_tensor_B)))};
    auto thread_layout_C_predicate_tensor_C{cute::make_tensor<bool>(
        cute::make_shape(cute::size<0>(thread_layout_C_global_block_tensor_C),
                         cute::size<1>(thread_layout_C_global_block_tensor_C)),
        cute::make_stride(
            cute::Int<1>{},
            cute::size<0>(thread_layout_C_global_block_tensor_C)))};
    // Create identity tensors.
    auto identity_tensor_A{cute::make_identity_tensor(cute::make_shape(
        cute::size<0>(smem_tensor_A), cute::size<1>(smem_tensor_A)))};
    auto identity_tensor_B{cute::make_identity_tensor(cute::make_shape(
        cute::size<0>(smem_tensor_B), cute::size<1>(smem_tensor_B)))};
    auto identity_tensor_C{cute::make_identity_tensor(
        cute::make_shape(cute::size<0>(global_block_tensor_C),
                         cute::size<1>(global_block_tensor_C)))};
    auto thread_layout_A_identity_tensor_A{
        thread_copy_A.partition_S(identity_tensor_A)}; // (CPY, CPY_M, CPY_K)
    auto thread_layout_B_identity_tensor_B{
        thread_copy_B.partition_S(identity_tensor_B)}; // (CPY, CPY_N, CPY_K)
    auto thread_layout_C_identity_tensor_C{
        cute::local_partition(identity_tensor_C, thread_layout_C,
                              threadIdx.x)}; // (BLK_M / THR_M, BLK_N / THR_N)

    CUTE_UNROLL
    for (auto m{0}; m < cute::size<0>(thread_layout_A_predicate_tensor_A); ++m)
    {
        for (auto k{0}; k < cute::size<1>(thread_layout_A_predicate_tensor_A);
             ++k)
        {
            thread_layout_A_predicate_tensor_A(m, k) =
                cute::get<0>(thread_layout_A_identity_tensor_A(0, m, k)) +
                        blockIdx.x * cute::size<0>(smem_tensor_A) <
                    cute::size<0>(shape_MNK) &&
                cute::get<1>(thread_layout_A_identity_tensor_A(0, m, k)) +
                        blockIdx.y * cute::size<1>(smem_tensor_A) <
                    cute::size<2>(shape_MNK);
        }
    }
    CUTE_UNROLL
    for (auto n{0}; n < cute::size<0>(thread_layout_B_predicate_tensor_B); ++n)
    {
        for (auto k{0}; k < cute::size<1>(thread_layout_B_predicate_tensor_B);
             ++k)
        {
            thread_layout_B_predicate_tensor_B(n, k) =
                cute::get<0>(thread_layout_B_identity_tensor_B(0, n, k)) +
                        blockIdx.y * cute::size<0>(smem_tensor_B) <
                    cute::size<1>(shape_MNK) &&
                cute::get<1>(thread_layout_B_identity_tensor_B(0, n, k)) +
                        blockIdx.x * cute::size<1>(smem_tensor_B) <
                    cute::size<2>(shape_MNK);
        }
    }
    CUTE_UNROLL
    for (auto m{0}; m < cute::size<0>(thread_layout_C_predicate_tensor_C); ++m)
    {
        CUTE_UNROLL
        for (auto n{0}; n < cute::size<1>(thread_layout_C_predicate_tensor_C);
             ++n)
        {
            thread_layout_C_predicate_tensor_C(m, n) =
                cute::get<0>(thread_layout_C_identity_tensor_C(m, n)) +
                        blockIdx.x * cute::size<0>(global_block_tensor_C) <
                    cute::size<0>(shape_MNK) &&
                cute::get<1>(thread_layout_C_identity_tensor_C(m, n)) +
                        blockIdx.y * cute::size<1>(global_block_tensor_C) <
                    cute::size<1>(shape_MNK);
        }
    }

    // Perform the gemm computation loop.
    auto const num_tiles_k{cute::size<2>(global_block_tensor_A)}; // k

    for (auto tile_idx_k{0}; tile_idx_k < num_tiles_k; ++tile_idx_k)
    {
        // Clear the shared memory buffers.
        // This is necessary when predicates are used for copying data from
        // global memory to shared memory so that mma will not be affected by
        // the previous data in the unwanted region.
        cute::clear(thread_layout_A_smem_tensor_A);
        cute::clear(thread_layout_B_smem_tensor_B);

        cute::copy_if(copy_A, thread_layout_A_predicate_tensor_A,
                      thread_layout_A_global_block_tensor_A(
                          cute::_, cute::_, cute::_, tile_idx_k),
                      thread_layout_A_smem_tensor_A);
        cute::copy_if(copy_B, thread_layout_B_predicate_tensor_B,
                      thread_layout_B_global_block_tensor_B(
                          cute::_, cute::_, cute::_, tile_idx_k),
                      thread_layout_B_smem_tensor_B);

        // Synchronize the threads to ensure the data copy is completed.
        cute::cp_async_fence();
        cute::cp_async_wait<0>();
        __syncthreads();

        // Compute gemm on thread_layout_C thread-partitioned smem.
        // This implicitly uses the UniversalFMA GEMM atom.
        cute::gemm(thread_layout_C_smem_tensor_A, thread_layout_C_smem_tensor_B,
                   thread_layout_C_register_tensor_C); // (BLK_M / THR_M, BLK_N
                                                       // / THR_N) += (BLK_M /
                                                       // THR_M, BLK_K) * (BLK_N
                                                       // / THR_N, BLK_K)

        __syncthreads();
    }

    // Scale and accumulate the result from the register tensor to the global
    // block tensor.
    cute::axpby(alpha, thread_layout_C_register_tensor_C, beta,
                thread_layout_C_global_block_tensor_C,
                thread_layout_C_predicate_tensor_C);
}

template <class TA, class TB, class TC, class Alpha, class Beta, class AStride,
          class BStride, class CStride, class VectorTypeA, class VectorTypeB>
static hipError_t gemm_base_tiled_copy(int m, int n, int k, Alpha alpha,
                                        TA const* A, int ldA, TB const* B,
                                        int ldB, Beta beta, TC* C, int ldC,
                                        AStride stride_A, BStride stride_B,
                                        CStride stride_C, hipStream_t stream)
{
    // Define GEMM shape.
    auto const M{m};
    auto const N{n};
    auto const K{k};
    auto const gemm_shape{cute::make_shape(M, N, K)}; // (M, N, K)

    // Define CTA size.
    auto const bM{cute::Int<128 * 4 / sizeof(TA)>{}};
    auto const bN{cute::Int<128 * 4 / sizeof(TB)>{}};
    auto const bK{cute::Int<8>{}};
    auto const cta_tiler{cute::make_shape(bM, bN, bK)}; // (BLK_M, BLK_N, BLK_K)

    // Define smem layouts.
    // smem_layout_A is (BLK_M, BLK_K) column-major.
    // smem_layout_B is (BLK_N, BLK_K) column-major.
    // smem_layout_C is (BLK_M, BLK_N) column-major.
    auto const smem_shape_A{cute::make_shape(bM, bK)}; // (BLK_M, BLK_K)
    auto const smem_stride_A{cute::make_stride(
        cute::Int<1>{}, cute::size<0>(smem_shape_A))}; // column-major
    auto const smem_layout_A{
        cute::make_layout(smem_shape_A, smem_stride_A)}; // (BLK_M, BLK_K)
    auto const smem_shape_B{cute::make_shape(bN, bK)};   // (BLK_N, BLK_K)
    auto const smem_stride_B{cute::make_stride(
        cute::Int<1>{}, cute::size<0>(smem_shape_B))}; // column-major
    auto const smem_layout_B{
        cute::make_layout(smem_shape_B, smem_stride_B)}; // (BLK_N, BLK_K)
    auto const smem_shape_C{cute::make_shape(bM, bN)};   // (BLK_M, BLK_N)
    auto const smem_stride_C{cute::make_stride(
        cute::Int<1>{}, cute::size<0>(smem_shape_C))}; // column-major
    auto const smem_layout_C{
        cute::make_layout(smem_shape_C, smem_stride_C)}; // (BLK_M, BLK_N)

    // Define thread layouts.
    auto const thread_shape_A{
        cute::make_shape(cute::Int<32>{}, cute::Int<8>{})}; // (THR_M, THR_K)
    auto const thread_shape_B{
        cute::make_shape(cute::Int<32>{}, cute::Int<8>{})}; // (THR_N, THR_K)
    auto const thread_shape_C{
        cute::make_shape(cute::Int<16>{}, cute::Int<16>{})}; // (THR_M, THR_N)
    auto const thread_stride_A{cute::make_stride(
        cute::Int<1>{}, cute::size<0>(thread_shape_A))}; // column-major
    auto const thread_stride_B{cute::make_stride(
        cute::Int<1>{}, cute::size<0>(thread_shape_B))}; // column-major
    auto const thread_stride_C{cute::make_stride(
        cute::Int<1>{}, cute::size<0>(thread_shape_C))}; // column-major
    auto const thread_layout_A{
        cute::make_layout(thread_shape_A, thread_stride_A)}; // (THR_M, THR_K)
    auto const thread_layout_B{
        cute::make_layout(thread_shape_B, thread_stride_B)}; // (THR_N, THR_K)
    auto const thread_layout_C{
        cute::make_layout(thread_shape_C, thread_stride_C)}; // (THR_M, THR_N)
    CUTE_STATIC_ASSERT_V(cute::size(thread_layout_A) ==
                         cute::size(thread_layout_B));
    CUTE_STATIC_ASSERT_V(cute::size(thread_layout_A) ==
                         cute::size(thread_layout_C));

    // CTA tiler has to be divisible by the thread layouts.
    CUTE_STATIC_ASSERT_V(cute::size<0>(cta_tiler) %
                             cute::size<0>(thread_layout_A) ==
                         cute::Int<0>{}); // BLK_M % THR_M == 0
    CUTE_STATIC_ASSERT_V(cute::size<2>(cta_tiler) %
                             cute::size<1>(thread_layout_A) ==
                         cute::Int<0>{}); // BLK_K % THR_K == 0
    CUTE_STATIC_ASSERT_V(cute::size<1>(cta_tiler) %
                             cute::size<0>(thread_layout_B) ==
                         cute::Int<0>{}); // BLK_N % THR_N == 0
    CUTE_STATIC_ASSERT_V(cute::size<2>(cta_tiler) %
                             cute::size<1>(thread_layout_B) ==
                         cute::Int<0>{}); // BLK_K % THR_K == 0
    CUTE_STATIC_ASSERT_V(cute::size<0>(cta_tiler) %
                             cute::size<0>(thread_layout_C) ==
                         cute::Int<0>{}); // BLK_M % THR_M == 0
    CUTE_STATIC_ASSERT_V(cute::size<1>(cta_tiler) %
                             cute::size<1>(thread_layout_C) ==
                         cute::Int<0>{}); // BLK_N % THR_N == 0

    // Shared memory layouts have to be divisible by the thread layouts.
    CUTE_STATIC_ASSERT_V(cute::size<0>(smem_layout_A) %
                             cute::size<0>(thread_layout_A) ==
                         cute::Int<0>{}); // BLK_M % THR_M == 0
    CUTE_STATIC_ASSERT_V(cute::size<1>(smem_layout_A) %
                             cute::size<1>(thread_layout_A) ==
                         cute::Int<0>{}); // BLK_K % THR_K == 0
    CUTE_STATIC_ASSERT_V(cute::size<0>(smem_layout_B) %
                             cute::size<0>(thread_layout_B) ==
                         cute::Int<0>{}); // BLK_N % THR_N == 0
    CUTE_STATIC_ASSERT_V(cute::size<1>(smem_layout_B) %
                             cute::size<1>(thread_layout_B) ==
                         cute::Int<0>{}); // BLK_K % THR_K == 0

    constexpr auto NUM_VECTOR_ELEMENTS_A{sizeof(VectorTypeA) / sizeof(TA)};
    auto const vector_shape_A{
        cute::make_shape(cute::Int<NUM_VECTOR_ELEMENTS_A>{},
                         cute::Int<1>{})}; // (NUM_VECTOR_ELEMENTS_A, 1)
    auto const vector_stride_A{cute::make_stride(
        cute::Int<1>{}, cute::size<0>(vector_shape_A))}; // column-major
    auto const vector_layout_A{cute::make_layout(
        vector_shape_A, vector_stride_A)}; // (NUM_VECTOR_ELEMENTS_A, 1)
    auto copy_A{cute::make_tiled_copy(
        cute::Copy_Atom<cute::UniversalCopy<VectorTypeA>, TA>{},
        thread_layout_A,
        vector_layout_A)}; // Thread layout: (THR_M, THR_K) Value layout:
                           // (NUM_VECTOR_ELEMENTS_A, 1)
    CUTE_STATIC_ASSERT_V(
        cute::size<0>(smem_layout_A) %
            (cute::size<0>(thread_layout_A) * cute::size<0>(vector_layout_A)) ==
        cute::Int<0>{}); // BLK_M % (THR_M * NUM_VECTOR_ELEMENTS_A) == 0

    auto const NUM_VECTOR_ELEMENTS_B{sizeof(VectorTypeB) / sizeof(TB)};
    auto const vector_shape_B{
        cute::make_shape(cute::Int<NUM_VECTOR_ELEMENTS_B>{},
                         cute::Int<1>{})}; // (NUM_VECTOR_ELEMENTS_B, 1)
    auto const vector_stride_B{cute::make_stride(
        cute::Int<1>{}, cute::size<0>(vector_shape_B))}; // column-major
    auto const vector_layout_B{cute::make_layout(
        vector_shape_B, vector_stride_B)}; // (NUM_VECTOR_ELEMENTS_B, 1)
    auto copy_B{cute::make_tiled_copy(
        cute::Copy_Atom<cute::UniversalCopy<VectorTypeB>, TB>{},
        thread_layout_B,
        vector_layout_B)}; // Thread layout: (THR_N, THR_K) Value layout:
                           // (NUM_VECTOR_ELEMENTS_B, 1)
    CUTE_STATIC_ASSERT_V(
        cute::size<0>(smem_layout_B) %
            (cute::size<0>(thread_layout_B) * cute::size<0>(vector_layout_B)) ==
        cute::Int<0>{}); // BLK_N % (THR_N * NUM_VECTOR_ELEMENTS_B) == 0

    // Swizzle parameters.
    constexpr int NUM_SHIFT_BITS_A{constexpr_log2(bM)};
    constexpr int NUM_MASK_BITS_A{constexpr_log2(32)};
    constexpr int NUM_BASE_BITS_A{constexpr_log2(NUM_VECTOR_ELEMENTS_A)};

    constexpr int NUM_SHIFT_BITS_B{constexpr_log2(bN)};
    constexpr int NUM_MASK_BITS_B{constexpr_log2(32)};
    constexpr int NUM_BASE_BITS_B{constexpr_log2(NUM_VECTOR_ELEMENTS_B)};

    auto const swizzle_A{
        cute::Swizzle<NUM_MASK_BITS_A, NUM_BASE_BITS_A, NUM_SHIFT_BITS_A>{}};
    auto const swizzle_B{
        cute::Swizzle<NUM_MASK_BITS_B, NUM_BASE_BITS_B, NUM_SHIFT_BITS_B>{}};

    // In fact, for some layouts, swizzles are not needed if no transpose is
    // performed.
    // But it should not reduce the performance even if the transpose is not
    // performed.
    auto const smem_layout_A_swizzled{
        cute::composition(swizzle_A, smem_layout_A)};
    auto const smem_layout_B_swizzled{
        cute::composition(swizzle_B, smem_layout_B)};

    // Launch the kernel.
    dim3 const block_dims{
        static_cast<unsigned int>(cute::size(thread_layout_C))};
    dim3 const grid_dims{
        static_cast<unsigned int>(cute::size(cute::ceil_div(M, bM))),
        static_cast<unsigned int>(cute::size(cute::ceil_div(N, bN)))};
    general_matrix_multiplication_naive_tiled_copy<<<grid_dims, block_dims, 0,
                                                     stream>>>(
        gemm_shape, cta_tiler, A, stride_A, smem_layout_A_swizzled,
        thread_layout_A, copy_A, B, stride_B, smem_layout_B_swizzled,
        thread_layout_B, copy_B, C, stride_C, smem_layout_C, thread_layout_C,
        alpha, beta);

    return hipGetLastError();
}

// The shape of A is (M, K) and the shape of B is (K, N).
// Then A is (M, K) column-major and B is (K, N) column-major.
// Then A is (M, K) column-major and B is (N, K) row-major.
template <class TA, class TB, class TC, class Alpha, class Beta>
static hipError_t gemm_nn(int m, int n, int k, Alpha alpha, TA const* A,
                           int ldA, TB const* B, int ldB, Beta beta, TC* C,
                           int ldC, hipStream_t stream)
{
    // Define global memory layouts.
    // A is (M, K) column-major.
    auto const stride_A{cute::make_stride(cute::Int<1>{}, ldA)}; // column-major
    // B is (N, K) row-major.
    auto const stride_B{cute::make_stride(ldB, cute::Int<1>{})}; // row-major
    // C is (M, N) column-major.
    auto const stride_C{cute::make_stride(cute::Int<1>{}, ldC)}; // column-major

    using VectorTypeA = cute::uint128_t;
    using VectorTypeB = TB;

    return gemm_base_tiled_copy<TA, TB, TC, Alpha, Beta, decltype(stride_A),
                                decltype(stride_B), decltype(stride_C),
                                VectorTypeA, VectorTypeB>(
        m, n, k, alpha, A, ldA, B, ldB, beta, C, ldC, stride_A, stride_B,
        stride_C, stream);
}

// The shape of A is (M, K) and the shape of transposed B is (K, N).
// Then A is (M, K) column-major and B is (N, K) column-major.
// The smem_A is (BLK_M, BLK_K) column-major and smem_B is (BLK_N, BLK_K)
// column-major.
template <class TA, class TB, class TC, class Alpha, class Beta>
static hipError_t gemm_nt(int m, int n, int k, Alpha alpha, TA const* A,
                           int ldA, TB const* B, int ldB, Beta beta, TC* C,
                           int ldC, hipStream_t stream)
{
    // Define global memory layouts.
    // A is (M, K) column-major.
    auto const stride_A{cute::make_stride(cute::Int<1>{}, ldA)}; // column-major
    // B is (N, K) column-major.
    auto const stride_B{cute::make_stride(cute::Int<1>{}, ldB)}; // column-major
    // C is (M, N) column-major.
    auto const stride_C{cute::make_stride(cute::Int<1>{}, ldC)}; // column-major

    using VectorTypeA = cute::uint128_t;
    using VectorTypeB = cute::uint128_t;

    return gemm_base_tiled_copy<TA, TB, TC, Alpha, Beta, decltype(stride_A),
                                decltype(stride_B), decltype(stride_C),
                                VectorTypeA, VectorTypeB>(
        m, n, k, alpha, A, ldA, B, ldB, beta, C, ldC, stride_A, stride_B,
        stride_C, stream);
}

// The shape of transposed A is (M, K) and the shape of B is (K, N).
// Then A is (K, M) column-major and B is (K, N) column-major.
// Then A is (M, K) row-major and B is (N, K) row-major.
template <class TA, class TB, class TC, class Alpha, class Beta>
static hipError_t gemm_tn(int m, int n, int k, Alpha alpha, TA const* A,
                           int ldA, TB const* B, int ldB, Beta beta, TC* C,
                           int ldC, hipStream_t stream)
{
    // Define global memory layouts.
    // A is (M, K) row-major.
    auto const stride_A{cute::make_stride(ldA, cute::Int<1>{})}; // row-major
    // B is (N, K) row-major.
    auto const stride_B{cute::make_stride(ldB, cute::Int<1>{})}; // row-major
    // C is (M, N) column-major.
    auto const stride_C{cute::make_stride(cute::Int<1>{}, ldC)}; // column-major

    // Because the shared memory layout is (BLK_M, BLK_K) column-major and
    // the global memory layout is (M, K) row-major, a transpose is needed and
    // vectorized memory copy is not possible. This transpose will result in
    // shared memory bank conflicts if not padding or swizzling is used. Another
    // strategy is to make the shared memory layout (BLK_M, BLK_K) row-major and
    // then we could perform vectorized memory copy. However, even with
    // swizzling or padding, there are can still be shared memory bank
    // conflicts. See https://leimao.github.io/blog/CuTe-Swizzle/ for more
    // information. So it is matter of experimentation to find the best strategy
    // for a specific problem. For this example, we will use the first strategy
    // without thoroughly investigating which strategy is better.
    using VectorTypeA = TA;
    using VectorTypeB = TB;

    return gemm_base_tiled_copy<TA, TB, TC, Alpha, Beta, decltype(stride_A),
                                decltype(stride_B), decltype(stride_C),
                                VectorTypeA, VectorTypeB>(
        m, n, k, alpha, A, ldA, B, ldB, beta, C, ldC, stride_A, stride_B,
        stride_C, stream);
}

// The shape of transposed A is (M, K) and the shape of transposed B is (K, N).
//    Then A is (K, M) column-major and B is (N, K) column-major.
//    Then A is (M, K) row-major and B is (N, K) column-major.
template <class TA, class TB, class TC, class Alpha, class Beta>
static hipError_t gemm_tt(int m, int n, int k, Alpha alpha, TA const* A,
                           int ldA, TB const* B, int ldB, Beta beta, TC* C,
                           int ldC, hipStream_t stream)
{
    // Define global memory layouts.
    // A is (M, K) row-major.
    auto const stride_A{cute::make_stride(ldA, cute::Int<1>{})}; // row-major
    // B is (N, K) column-major.
    auto const stride_B{cute::make_stride(cute::Int<1>{}, ldB)}; // column-major
    // C is (M, N) column-major.
    auto const stride_C{cute::make_stride(cute::Int<1>{}, ldC)}; // column-major

    using VectorTypeA = TA;
    using VectorTypeB = cute::uint128_t;

    return gemm_base_tiled_copy<TA, TB, TC, Alpha, Beta, decltype(stride_A),
                                decltype(stride_B), decltype(stride_C),
                                VectorTypeA, VectorTypeB>(
        m, n, k, alpha, A, ldA, B, ldB, beta, C, ldC, stride_A, stride_B,
        stride_C, stream);
}

template <class TA, class TB, class TC, class Alpha, class Beta>
hipError_t launch_gemm_naive_tiled_copy(char transA, char transB, int m, int n,
                                         int k, Alpha alpha, TA const* A,
                                         int ldA, TB const* B, int ldB,
                                         Beta beta, TC* C, int ldC,
                                         hipStream_t stream)
{
    // To ensure vectorized memory access, the values of m, n, and k are
    // constrained to be:
    if (m * sizeof(TA) % 128 != 0 || k * sizeof(TA) % 128 != 0)
    {
        return hipErrorNotSupported;
    }
    if (k * sizeof(TB) % 128 != 0 || n * sizeof(TB) % 128 != 0)
    {
        return hipErrorNotSupported;
    }
    if (m * sizeof(TC) % 128 != 0 || n * sizeof(TC) % 128 != 0)
    {
        return hipErrorNotSupported;
    }
    // To ensure data alignment, the values of ldA, ldB, and ldC are constrained
    // to be:
    if (ldA * sizeof(TA) % 128 != 0 || ldB * sizeof(TB) % 128 != 0 ||
        ldC * sizeof(TC) % 128 != 0)
    {
        return hipErrorNotSupported;
    }

    if (transA == 'N' && transB == 'T')
    {
        return gemm_nt(m, n, k, alpha, A, ldA, B, ldB, beta, C, ldC, stream);
    }
    else if (transA == 'N' && transB == 'N')
    {
        return gemm_nn(m, n, k, alpha, A, ldA, B, ldB, beta, C, ldC, stream);
    }
    else if (transA == 'T' && transB == 'N')
    {
        return gemm_tn(m, n, k, alpha, A, ldA, B, ldB, beta, C, ldC, stream);
    }
    else if (transA == 'T' && transB == 'T')
    {
        return gemm_tt(m, n, k, alpha, A, ldA, B, ldB, beta, C, ldC, stream);
    }
    else
    {
        return hipErrorNotSupported;
    }
}

// Explicit instantiation
template hipError_t
launch_gemm_naive_tiled_copy<float, float, float, float, float>(
    char transA, char transB, int m, int n, int k, float alpha, float const* A,
    int ldA, float const* B, int ldB, float beta, float* C, int ldC,
    hipStream_t stream);
template hipError_t
launch_gemm_naive_tiled_copy<double, double, double, double, double>(
    char transA, char transB, int m, int n, int k, double alpha,
    double const* A, int ldA, double const* B, int ldB, double beta, double* C,
    int ldC, hipStream_t stream);
template hipError_t launch_gemm_naive_tiled_copy<cute::half_t, cute::half_t,
                                                  cute::half_t, float, float>(
    char transA, char transB, int m, int n, int k, float alpha,
    cute::half_t const* A, int ldA, cute::half_t const* B, int ldB, float beta,
    cute::half_t* C, int ldC, hipStream_t stream);
template hipError_t
launch_gemm_naive_tiled_copy<cute::half_t, cute::half_t, cute::half_t,
                             cute::half_t, cute::half_t>(
    char transA, char transB, int m, int n, int k, cute::half_t alpha,
    cute::half_t const* A, int ldA, cute::half_t const* B, int ldB,
    cute::half_t beta, cute::half_t* C, int ldC, hipStream_t stream);