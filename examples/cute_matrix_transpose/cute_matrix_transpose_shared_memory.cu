#include "hip/hip_runtime.h"
#include <iomanip>
#include <iostream>

#include <hip/hip_runtime.h>

#include <cute/tensor.hpp>

#include "cute_matrix_transpose.hpp"

template <class TensorSrc, class TensorDst, class SharedMemoryLayoutSrc,
          class SharedMemoryLayoutDst, class ThreadLayoutSrc,
          class ThreadLayoutDst>
__global__ void static matrix_transpose_shared_memory(
    TensorSrc tensor_src, TensorDst tensor_dst, SharedMemoryLayoutSrc,
    SharedMemoryLayoutDst, ThreadLayoutSrc, ThreadLayoutDst)
{
    using Element = typename TensorSrc::value_type;
    CUTE_STATIC_ASSERT(cute::size(SharedMemoryLayoutSrc{}) ==
                           cute::size(SharedMemoryLayoutDst{}),
                       "SharedMemoryLayoutSrc and SharedMemoryLayoutDst "
                       "must have the same size.");
    __shared__ Element shared_memory[cute::cosize(SharedMemoryLayoutSrc{})];

    auto tensor_cache_src{cute::make_tensor(cute::make_smem_ptr(shared_memory),
                                            SharedMemoryLayoutSrc{})};
    auto tensor_cache_dst{cute::make_tensor(cute::make_smem_ptr(shared_memory),
                                            SharedMemoryLayoutDst{})};

    auto global_tile_src{tensor_src(cute::make_coord(cute::_, cute::_),
                                    blockIdx.y,
                                    blockIdx.x)}; // (TileSizeY, TileSizeX)
    auto global_tile_dst{tensor_dst(cute::make_coord(cute::_, cute::_),
                                    blockIdx.y,
                                    blockIdx.x)}; // (TileSizeY, TileSizeX)

    auto thread_global_tile_src{cute::local_partition(
        global_tile_src, ThreadLayoutSrc{},
        threadIdx.x)}; // (ThreadValueSizeY, ThreadValueSizeX)
    auto thread_global_tile_dst{cute::local_partition(
        global_tile_dst, ThreadLayoutDst{},
        threadIdx.x)}; // (ThreadValueSizeX, ThreadValueSizeY)

    auto thread_shared_tile_src{cute::local_partition(
        tensor_cache_src, ThreadLayoutSrc{},
        threadIdx.x)}; // (ThreadValueSizeY, ThreadValueSizeX)
    auto thread_shared_tile_dst{cute::local_partition(
        tensor_cache_dst, ThreadLayoutDst{},
        threadIdx.x)}; // (ThreadValueSizeX, ThreadValueSizeY)

    // A 2D array of tuples that maps (x, y) to (x, y).
    auto const identity_tensor_src{cute::make_identity_tensor(cute::make_shape(
        cute::size<0>(global_tile_src), cute::size<1>(global_tile_src)))};
    auto const thread_identity_tensor_src{cute::local_partition(
        identity_tensor_src, ThreadLayoutSrc{}, threadIdx.x)};
    auto predicator_src{cute::make_tensor<bool>(
        cute::make_shape(cute::size<0>(thread_global_tile_src),
                         cute::size<1>(thread_global_tile_src)))};

    auto const identity_tensor_dst{cute::make_identity_tensor(cute::make_shape(
        cute::size<0>(global_tile_dst), cute::size<1>(global_tile_dst)))};
    auto const thread_identity_tensor_dst{cute::local_partition(
        identity_tensor_dst, ThreadLayoutDst{}, threadIdx.x)};
    auto predicator_dst{cute::make_tensor<bool>(
        cute::make_shape(cute::size<0>(thread_global_tile_dst),
                         cute::size<1>(thread_global_tile_dst)))};

    auto const num_max_columns{cute::stride<0>(global_tile_src)};
    auto const num_max_rows{cute::stride<1>(global_tile_dst)};
    constexpr auto global_tile_columns{cute::size<1>(global_tile_src)};
    constexpr auto global_tile_rows{cute::size<0>(global_tile_src)};

    CUTE_UNROLL
    for (unsigned int i{0}; i < cute::size<0>(predicator_src); ++i)
    {
        CUTE_UNROLL
        for (unsigned int j{0}; j < cute::size<1>(predicator_src); ++j)
        {
            auto const thread_identity{thread_identity_tensor_src(i, j)};
            bool const is_row_in_bound{cute::get<0>(thread_identity) +
                                           blockIdx.y * global_tile_rows <
                                       num_max_rows};
            bool const is_column_in_bound{cute::get<1>(thread_identity) +
                                              blockIdx.x * global_tile_columns <
                                          num_max_columns};
            predicator_src(i, j) = is_row_in_bound && is_column_in_bound;
        }
    }

    CUTE_UNROLL
    for (unsigned int i{0}; i < cute::size<0>(predicator_dst); ++i)
    {
        CUTE_UNROLL
        for (unsigned int j{0}; j < cute::size<1>(predicator_dst); ++j)
        {
            auto const thread_identity{thread_identity_tensor_dst(i, j)};
            bool const is_row_in_bound{cute::get<0>(thread_identity) +
                                           blockIdx.y * global_tile_rows <
                                       num_max_rows};
            bool const is_column_in_bound{cute::get<1>(thread_identity) +
                                              blockIdx.x * global_tile_columns <
                                          num_max_columns};
            predicator_dst(i, j) = is_row_in_bound && is_column_in_bound;
        }
    }

    cute::copy_if(predicator_src, thread_global_tile_src,
                  thread_shared_tile_src);
    cute::cp_async_fence();
    cute::cp_async_wait<0>();
    __syncthreads();
    cute::copy_if(predicator_dst, thread_shared_tile_dst,
                  thread_global_tile_dst);
}

template <class TensorSrc, class TensorDst, class SharedMemoryLayoutSrc,
          class SharedMemoryLayoutDst, class ThreadLayoutSrc,
          class ThreadLayoutDst, class VectorLayout>
__global__ void static matrix_transpose_shared_memory_vectorized(
    TensorSrc tensor_src, TensorDst tensor_dst, SharedMemoryLayoutSrc,
    SharedMemoryLayoutDst, ThreadLayoutSrc, ThreadLayoutDst, VectorLayout)
{
    using Element = typename TensorSrc::value_type;
    CUTE_STATIC_ASSERT(cute::size(SharedMemoryLayoutSrc{}) ==
                           cute::size(SharedMemoryLayoutDst{}),
                       "SharedMemoryLayoutSrc and SharedMemoryLayoutDst "
                       "must have the same size.");
    __shared__ Element shared_memory[cute::cosize(SharedMemoryLayoutSrc{})];

    auto tensor_cache_src{cute::make_tensor(cute::make_smem_ptr(shared_memory),
                                            SharedMemoryLayoutSrc{})};
    auto tensor_cache_dst{cute::make_tensor(cute::make_smem_ptr(shared_memory),
                                            SharedMemoryLayoutDst{})};

    auto global_tile_src{tensor_src(cute::make_coord(cute::_, cute::_),
                                    blockIdx.y,
                                    blockIdx.x)}; // (TileSizeY, TileSizeX)
    auto global_tile_dst{tensor_dst(cute::make_coord(cute::_, cute::_),
                                    blockIdx.y,
                                    blockIdx.x)}; // (TileSizeY, TileSizeX)

    using AccessType =
        cutlass::AlignedArray<Element, cute::size(VectorLayout{})>;
    using CopyAtom = cute::Copy_Atom<cute::UniversalCopy<AccessType>, Element>;
    auto tiled_copy_src{
        cute::make_tiled_copy(CopyAtom{}, ThreadLayoutSrc{}, VectorLayout{})};
    auto thread_copy_src{tiled_copy_src.get_thread_slice(threadIdx.x)};

    auto thread_global_tile_src{thread_copy_src.partition_S(
        global_tile_src)}; // (CopyAtomShape, NumCopyTile)
    auto thread_shared_tile_src{thread_copy_src.partition_D(
        tensor_cache_src)}; // (CopyAtomShape, NumCopyTile)

    auto thread_global_tile_dst{cute::local_partition(
        global_tile_dst, ThreadLayoutDst{},
        threadIdx.x)}; // (ThreadValueSizeX, ThreadValueSizeY)
    auto thread_shared_tile_dst{cute::local_partition(
        tensor_cache_dst, ThreadLayoutDst{},
        threadIdx.x)}; // (ThreadValueSizeX, ThreadValueSizeY)

    auto const num_max_columns{cute::stride<0>(global_tile_src)};
    auto const num_max_rows{cute::stride<1>(global_tile_dst)};
    constexpr auto global_tile_columns{cute::size<1>(global_tile_src)};
    constexpr auto global_tile_rows{cute::size<0>(global_tile_src)};

    // A 2D array of tuples that maps (x, y) to (x, y).
    auto const identity_tensor_src{cute::make_identity_tensor(cute::make_shape(
        cute::size<0>(global_tile_src), cute::size<1>(global_tile_src)))};
    auto thread_identity_tensor_src{thread_copy_src.partition_S(
        identity_tensor_src)}; // (CopyAtomShape, NumCopyTile)
    auto predicator_src{cute::make_tensor<bool>(
        cute::make_shape(cute::size<1>(thread_global_tile_src),
                         cute::size<2>(thread_global_tile_src)))};

    CUTE_UNROLL
    for (unsigned int i{0}; i < cute::size<0>(predicator_src); ++i)
    {
        CUTE_UNROLL
        for (unsigned int j{0}; j < cute::size<1>(predicator_src); ++j)
        {
            auto const thread_identity{thread_identity_tensor_src(0, i, j)};
            bool const is_row_in_bound{cute::get<0>(thread_identity) +
                                           blockIdx.y * global_tile_rows <
                                       num_max_rows};
            bool const is_column_in_bound{cute::get<1>(thread_identity) +
                                              blockIdx.x * global_tile_columns <
                                          num_max_columns};
            predicator_src(i, j) = is_row_in_bound && is_column_in_bound;
        }
    }

    auto const identity_tensor_dst{cute::make_identity_tensor(cute::make_shape(
        cute::size<0>(global_tile_dst), cute::size<1>(global_tile_dst)))};
    auto const thread_identity_tensor_dst{cute::local_partition(
        identity_tensor_dst, ThreadLayoutDst{}, threadIdx.x)};
    auto predicator_dst{cute::make_tensor<bool>(
        cute::make_shape(cute::size<0>(thread_global_tile_dst),
                         cute::size<1>(thread_global_tile_dst)))};

    CUTE_UNROLL
    for (unsigned int i{0}; i < cute::size<0>(predicator_dst); ++i)
    {
        CUTE_UNROLL
        for (unsigned int j{0}; j < cute::size<1>(predicator_dst); ++j)
        {
            auto const thread_identity{thread_identity_tensor_dst(i, j)};
            bool const is_row_in_bound{cute::get<0>(thread_identity) +
                                           blockIdx.y * global_tile_rows <
                                       num_max_rows};
            bool const is_column_in_bound{cute::get<1>(thread_identity) +
                                              blockIdx.x * global_tile_columns <
                                          num_max_columns};
            predicator_dst(i, j) = is_row_in_bound && is_column_in_bound;
        }
    }

    cute::copy_if(tiled_copy_src, predicator_src, thread_global_tile_src,
                  thread_shared_tile_src);
    cute::cp_async_fence();
    cute::cp_async_wait<0>();
    __syncthreads();
    cute::copy_if(predicator_dst, thread_shared_tile_dst,
                  thread_global_tile_dst);
}

enum class SharedMemoryBankConflictAccessMode
{
    Read,
    Write
};

template <typename T>
static hipError_t launch_matrix_transpose_shared_memory_bank_conflict_base(
    T const* input_matrix, T* output_matrix, unsigned int M, unsigned int N,
    SharedMemoryBankConflictAccessMode bank_conflict_access_mode,
    hipStream_t stream)
{
    auto const tensor_shape{cute::make_shape(M, N)};
    auto const tensor_shape_transposed{cute::make_shape(N, M)};

    // Input matrix: row-major M x N matrix.
    auto const global_memory_layout_src{cute::make_layout(
        tensor_shape, cute::GenRowMajor{})}; // (M, N) : (N, 1)
    // Output matrix: row-major N x M matrix.
    auto const global_memory_layout_dst{cute::make_layout(
        tensor_shape_transposed, cute::GenRowMajor{})}; // (N, M) : (M, 1)
    // Same output matrix, but different view: column-major M x N matrix.
    auto const global_memory_layout_dst_transposed{cute::make_layout(
        tensor_shape, cute::GenColMajor{})}; // (M, N) : (1, M)

    auto const tensor_src{cute::make_tensor(cute::make_gmem_ptr(input_matrix),
                                            global_memory_layout_src)};
    auto const tensor_dst{cute::make_tensor(cute::make_gmem_ptr(output_matrix),
                                            global_memory_layout_dst)};
    auto const tensor_dst_transposed{
        cute::make_tensor(cute::make_gmem_ptr(output_matrix),
                          global_memory_layout_dst_transposed)};

    using TileSizeX = cute::Int<128>; // bN
    using TileSizeY = cute::Int<32>;  // bM

    constexpr auto block_shape{cute::make_shape(TileSizeY{}, TileSizeX{})};
    constexpr auto block_shape_transposed{
        cute::make_shape(TileSizeX{}, TileSizeY{})};

    auto const shared_memory_layout_src{cute::make_layout(
        block_shape, cute::GenRowMajor{})}; // (bM, bN) : (bN, 1)
    auto const shared_memory_layout_dst{cute::make_layout(
        block_shape_transposed, cute::GenRowMajor{})}; // (bN, bM) : (bM, 1)
    auto const shared_memory_layout_dst_transposed{cute::make_layout(
        block_shape, cute::GenColMajor{})}; // (bM, bN) : (1, bM)

    auto const tiled_tensor_src{cute::tiled_divide(
        tensor_src, block_shape)}; // ((TileSizeY, TileSizeX), M /
                                   // TileSizeY, N / TileSizeX)
    auto const tiled_tensor_dst{cute::tiled_divide(
        tensor_dst, block_shape_transposed)}; // ((TileSizeX, TileSizeY), N
                                              // / TileSizeX, M / TileSizeY)
    auto const tiled_tensor_dst_transposed{cute::tiled_divide(
        tensor_dst_transposed, block_shape)}; // ((TileSizeY, TileSizeX), M
                                              // / TileSizeY, N / TileSizeX)

    using ThreadBlockSizeX = cute::Int<32>; // tN
    using ThreadBlockSizeY = cute::Int<8>;  // tM

    CUTE_STATIC_ASSERT(TileSizeX::value % ThreadBlockSizeX::value == 0,
                       "TileSizeX must be divisible by ThreadBlockSizeX");
    CUTE_STATIC_ASSERT(TileSizeY::value % ThreadBlockSizeY::value == 0,
                       "TileSizeY must be divisible by ThreadBlockSizeY");

    constexpr auto thread_block_shape{
        cute::make_shape(ThreadBlockSizeY{}, ThreadBlockSizeX{})};
    constexpr auto thread_block_shape_transposed{
        cute::make_shape(ThreadBlockSizeX{}, ThreadBlockSizeY{})};
    constexpr auto thread_layout{
        cute::make_layout(thread_block_shape, cute::GenRowMajor{})};
    constexpr auto thread_layout_transposed{
        cute::make_layout(thread_block_shape_transposed, cute::GenColMajor{})};

    dim3 const grid_dim{cute::size<2>(tiled_tensor_src),
                        cute::size<1>(tiled_tensor_src)};
    dim3 const thread_dim{ThreadBlockSizeX::value * ThreadBlockSizeY::value};

    if (bank_conflict_access_mode == SharedMemoryBankConflictAccessMode::Read)
    {
        matrix_transpose_shared_memory<<<grid_dim, thread_dim, 0, stream>>>(
            tiled_tensor_src, tiled_tensor_dst_transposed,
            shared_memory_layout_src, shared_memory_layout_src, thread_layout,
            thread_layout_transposed);
    }
    else
    {
        matrix_transpose_shared_memory<<<grid_dim, thread_dim, 0, stream>>>(
            tiled_tensor_src, tiled_tensor_dst_transposed,
            shared_memory_layout_dst_transposed,
            shared_memory_layout_dst_transposed, thread_layout,
            thread_layout_transposed);
    }

    return hipGetLastError();
}

template <typename T>
static hipError_t
launch_matrix_transpose_shared_memory_vectorized_bank_conflict_base(
    T const* input_matrix, T* output_matrix, unsigned int M, unsigned int N,
    SharedMemoryBankConflictAccessMode bank_conflict_access_mode,
    hipStream_t stream)
{
    using VectorType = cute::uint128_t;
    static_assert(sizeof(VectorType) % sizeof(T) == 0,
                  "sizeof(VectorType) must be a multiple of sizeof(T)");
    constexpr unsigned int NUM_VECTOR_ELEMENTS{sizeof(VectorType) / sizeof(T)};

    if (N % NUM_VECTOR_ELEMENTS != 0)
    {
        return hipErrorInvalidValue;
    }

    auto const tensor_shape{cute::make_shape(M, N)};
    auto const tensor_shape_transposed{cute::make_shape(N, M)};

    // Input matrix: row-major M x N matrix.
    auto const global_memory_layout_src{cute::make_layout(
        tensor_shape, cute::GenRowMajor{})}; // (M, N) : (N, 1)
    // Output matrix: row-major N x M matrix.
    auto const global_memory_layout_dst{cute::make_layout(
        tensor_shape_transposed, cute::GenRowMajor{})}; // (N, M) : (M, 1)
    // Same output matrix, but different view: column-major M x N matrix.
    auto const global_memory_layout_dst_transposed{cute::make_layout(
        tensor_shape, cute::GenColMajor{})}; // (M, N) : (1, M)

    auto const tensor_src{cute::make_tensor(cute::make_gmem_ptr(input_matrix),
                                            global_memory_layout_src)};
    auto const tensor_dst{cute::make_tensor(cute::make_gmem_ptr(output_matrix),
                                            global_memory_layout_dst)};
    auto const tensor_dst_transposed{
        cute::make_tensor(cute::make_gmem_ptr(output_matrix),
                          global_memory_layout_dst_transposed)};

    using TileSizeX = cute::Int<128>; // bN
    using TileSizeY = cute::Int<32>;  // bM

    constexpr auto block_shape{cute::make_shape(TileSizeY{}, TileSizeX{})};
    constexpr auto block_shape_transposed{
        cute::make_shape(TileSizeX{}, TileSizeY{})};

    auto const shared_memory_layout_src{cute::make_layout(
        block_shape, cute::GenRowMajor{})}; // (bM, bN) : (bN, 1)
    auto const shared_memory_layout_dst{cute::make_layout(
        block_shape_transposed, cute::GenRowMajor{})}; // (bN, bM) : (bM, 1)
    auto const shared_memory_layout_dst_transposed{cute::make_layout(
        block_shape, cute::GenColMajor{})}; // (bM, bN) : (1, bM)

    auto const tiled_tensor_src{cute::tiled_divide(
        tensor_src, block_shape)}; // ((TileSizeY, TileSizeX), M /
                                   // TileSizeY, N / TileSizeX)
    auto const tiled_tensor_dst{cute::tiled_divide(
        tensor_dst, block_shape_transposed)}; // ((TileSizeX, TileSizeY), N
                                              // / TileSizeX, M / TileSizeY)
    auto const tiled_tensor_dst_transposed{cute::tiled_divide(
        tensor_dst_transposed, block_shape)}; // ((TileSizeY, TileSizeX), M
                                              // / TileSizeY, N / TileSizeX)

    using ThreadBlockSizeX = cute::Int<32>; // tN
    using ThreadBlockSizeY = cute::Int<8>;  // tM

    CUTE_STATIC_ASSERT(TileSizeX::value % ThreadBlockSizeX::value == 0,
                       "TileSizeX must be divisible by ThreadBlockSizeX");
    CUTE_STATIC_ASSERT(TileSizeY::value % ThreadBlockSizeY::value == 0,
                       "TileSizeY must be divisible by ThreadBlockSizeY");

    constexpr auto thread_block_shape{
        cute::make_shape(ThreadBlockSizeY{}, ThreadBlockSizeX{})};
    constexpr auto thread_block_shape_transposed{
        cute::make_shape(ThreadBlockSizeX{}, ThreadBlockSizeY{})};
    constexpr auto thread_layout{
        cute::make_layout(thread_block_shape, cute::GenRowMajor{})};
    constexpr auto thread_layout_transposed{
        cute::make_layout(thread_block_shape_transposed, cute::GenColMajor{})};

    using VECTOR_SIZE_X = cute::Int<NUM_VECTOR_ELEMENTS>;
    constexpr auto vector_shape{
        cute::make_shape(cute::Int<1>{}, VECTOR_SIZE_X{})};
    // Copy atom vector layout.
    constexpr auto vector_layout{
        cute::make_layout(vector_shape, cute::GenRowMajor{})};

    dim3 const grid_dim{cute::size<2>(tiled_tensor_src),
                        cute::size<1>(tiled_tensor_src)};
    dim3 const thread_dim{ThreadBlockSizeX::value * ThreadBlockSizeY::value};

    if (bank_conflict_access_mode == SharedMemoryBankConflictAccessMode::Read)
    {
        matrix_transpose_shared_memory_vectorized<<<grid_dim, thread_dim, 0,
                                                    stream>>>(
            tiled_tensor_src, tiled_tensor_dst_transposed,
            shared_memory_layout_src, shared_memory_layout_src, thread_layout,
            thread_layout_transposed, vector_layout);
    }
    else
    {
        return hipErrorInvalidValue;
    }

    return hipGetLastError();
}

template <typename T>
hipError_t launch_matrix_transpose_shared_memory_bank_conflict_read(
    T const* input_matrix, T* output_matrix, unsigned int M, unsigned int N,
    hipStream_t stream)
{
    return launch_matrix_transpose_shared_memory_bank_conflict_base(
        input_matrix, output_matrix, M, N,
        SharedMemoryBankConflictAccessMode::Read, stream);
}

template <typename T>
hipError_t launch_matrix_transpose_shared_memory_bank_conflict_write(
    T const* input_matrix, T* output_matrix, unsigned int M, unsigned int N,
    hipStream_t stream)
{
    return launch_matrix_transpose_shared_memory_bank_conflict_base(
        input_matrix, output_matrix, M, N,
        SharedMemoryBankConflictAccessMode::Write, stream);
}

template <typename T>
hipError_t launch_matrix_transpose_shared_memory_vectorized_bank_conflict_read(
    T const* input_matrix, T* output_matrix, unsigned int M, unsigned int N,
    hipStream_t stream)
{
    return launch_matrix_transpose_shared_memory_vectorized_bank_conflict_base<
        T>(input_matrix, output_matrix, M, N,
           SharedMemoryBankConflictAccessMode::Read, stream);
}

template <typename T>
static hipError_t launch_matrix_transpose_shared_memory_padded(
    T const* input_matrix, T* output_matrix, unsigned int M, unsigned int N,
    hipStream_t stream)
{
    auto const tensor_shape{cute::make_shape(M, N)};
    auto const tensor_shape_transposed{cute::make_shape(N, M)};

    // Input matrix: row-major M x N matrix.
    auto const global_memory_layout_src{cute::make_layout(
        tensor_shape, cute::GenRowMajor{})}; // (M, N) : (N, 1)
    // Output matrix: row-major N x M matrix.
    auto const global_memory_layout_dst{cute::make_layout(
        tensor_shape_transposed, cute::GenRowMajor{})}; // (N, M) : (M, 1)
    // Same output matrix, but different view: column-major M x N matrix.
    auto const global_memory_layout_dst_transposed{cute::make_layout(
        tensor_shape, cute::GenColMajor{})}; // (M, N) : (1, M)

    auto const tensor_src{cute::make_tensor(cute::make_gmem_ptr(input_matrix),
                                            global_memory_layout_src)};
    auto const tensor_dst{cute::make_tensor(cute::make_gmem_ptr(output_matrix),
                                            global_memory_layout_dst)};
    auto const tensor_dst_transposed{
        cute::make_tensor(cute::make_gmem_ptr(output_matrix),
                          global_memory_layout_dst_transposed)};

    using TileSizeX = cute::Int<64>;          // bN
    using TILE_SIZE_X_PADDED = cute::Int<65>; // bN + 1
    using TileSizeY = cute::Int<32>;          // bM

    constexpr auto block_shape{cute::make_shape(TileSizeY{}, TileSizeX{})};
    constexpr auto block_shape_transposed{
        cute::make_shape(TileSizeX{}, TileSizeY{})};

    auto const shared_memory_layout_src{cute::make_layout(
        block_shape, cute::GenRowMajor{})}; // (bM, bN) : (bN, 1)
    auto const shared_memory_layout_src_padded{cute::make_layout(
        block_shape,
        cute::make_stride(TILE_SIZE_X_PADDED{},
                          cute::Int<1>{}))}; // (bM, bN) : (bN + 1, 1)
    auto const shared_memory_layout_dst{cute::make_layout(
        block_shape_transposed, cute::GenRowMajor{})}; // (bN, bM) : (bM, 1)
    auto const shared_memory_layout_dst_transposed{cute::make_layout(
        block_shape, cute::GenColMajor{})}; // (bM, bN) : (1, bM)

    auto const tiled_tensor_src{cute::tiled_divide(
        tensor_src, block_shape)}; // ((TileSizeY, TileSizeX), M /
                                   // TileSizeY, N / TileSizeX)
    auto const tiled_tensor_dst{cute::tiled_divide(
        tensor_dst, block_shape_transposed)}; // ((TileSizeX, TileSizeY), N
                                              // / TileSizeX, M / TileSizeY)
    auto const tiled_tensor_dst_transposed{cute::tiled_divide(
        tensor_dst_transposed, block_shape)}; // ((TileSizeY, TileSizeX), M
                                              // / TileSizeY, N / TileSizeX)

    using ThreadBlockSizeX = cute::Int<32>; // tN
    using ThreadBlockSizeY = cute::Int<8>;  // tM

    CUTE_STATIC_ASSERT(TileSizeX::value % ThreadBlockSizeX::value == 0,
                       "TileSizeX must be divisible by ThreadBlockSizeX");
    CUTE_STATIC_ASSERT(TileSizeY::value % ThreadBlockSizeY::value == 0,
                       "TileSizeY must be divisible by ThreadBlockSizeY");

    constexpr auto thread_block_shape{
        cute::make_shape(ThreadBlockSizeY{}, ThreadBlockSizeX{})};
    constexpr auto thread_block_shape_transposed{
        cute::make_shape(ThreadBlockSizeX{}, ThreadBlockSizeY{})};
    constexpr auto thread_layout{
        cute::make_layout(thread_block_shape, cute::GenRowMajor{})};
    constexpr auto thread_layout_transposed{
        cute::make_layout(thread_block_shape_transposed, cute::GenColMajor{})};

    dim3 const grid_dim{cute::size<2>(tiled_tensor_src),
                        cute::size<1>(tiled_tensor_src)};
    dim3 const thread_dim{ThreadBlockSizeX::value * ThreadBlockSizeY::value};

    matrix_transpose_shared_memory<<<grid_dim, thread_dim, 0, stream>>>(
        tiled_tensor_src, tiled_tensor_dst_transposed,
        shared_memory_layout_src_padded, shared_memory_layout_src_padded,
        thread_layout, thread_layout_transposed);

    return hipGetLastError();
}

template <typename T>
static hipError_t launch_matrix_transpose_shared_memory_vectorized_padded(
    T const* input_matrix, T* output_matrix, unsigned int M, unsigned int N,
    hipStream_t stream)
{
    using VectorType = cute::uint128_t;
    static_assert(sizeof(VectorType) % sizeof(T) == 0,
                  "sizeof(VectorType) must be a multiple of sizeof(T)");
    constexpr unsigned int NUM_VECTOR_ELEMENTS{sizeof(VectorType) / sizeof(T)};

    if (N % NUM_VECTOR_ELEMENTS != 0)
    {
        return hipErrorInvalidValue;
    }

    auto const tensor_shape{cute::make_shape(M, N)};
    auto const tensor_shape_transposed{cute::make_shape(N, M)};

    // Input matrix: row-major M x N matrix.
    auto const global_memory_layout_src{cute::make_layout(
        tensor_shape, cute::GenRowMajor{})}; // (M, N) : (N, 1)
    // Output matrix: row-major N x M matrix.
    auto const global_memory_layout_dst{cute::make_layout(
        tensor_shape_transposed, cute::GenRowMajor{})}; // (N, M) : (M, 1)
    // Same output matrix, but different view: column-major M x N matrix.
    auto const global_memory_layout_dst_transposed{cute::make_layout(
        tensor_shape, cute::GenColMajor{})}; // (M, N) : (1, M)

    auto const tensor_src{cute::make_tensor(cute::make_gmem_ptr(input_matrix),
                                            global_memory_layout_src)};
    auto const tensor_dst{cute::make_tensor(cute::make_gmem_ptr(output_matrix),
                                            global_memory_layout_dst)};
    auto const tensor_dst_transposed{
        cute::make_tensor(cute::make_gmem_ptr(output_matrix),
                          global_memory_layout_dst_transposed)};

    using TileSizeX = cute::Int<128>; // bN
    // Such padding is necessary for the byte alignment of the vectorized
    // access. However, the shared memory bank conflict mitigation can be
    // compromised.
    using TILE_SIZE_X_PADDED =
        cute::Int<128 + NUM_VECTOR_ELEMENTS>; // bN + NUM_VECTOR_ELEMENTS
    using TileSizeY = cute::Int<32>;          // bM

    constexpr auto block_shape{cute::make_shape(TileSizeY{}, TileSizeX{})};
    constexpr auto block_shape_transposed{
        cute::make_shape(TileSizeX{}, TileSizeY{})};

    auto const shared_memory_layout_src{cute::make_layout(
        block_shape, cute::GenRowMajor{})}; // (bM, bN) : (bN, 1)
    auto const shared_memory_layout_src_padded{cute::make_layout(
        block_shape,
        cute::make_stride(TILE_SIZE_X_PADDED{},
                          cute::Int<1>{}))}; // (bM, bN) : (bN + 1, 1)
    auto const shared_memory_layout_dst{cute::make_layout(
        block_shape_transposed, cute::GenRowMajor{})}; // (bN, bM) : (bM, 1)
    auto const shared_memory_layout_dst_transposed{cute::make_layout(
        block_shape, cute::GenColMajor{})}; // (bM, bN) : (1, bM)

    auto const tiled_tensor_src{cute::tiled_divide(
        tensor_src, block_shape)}; // ((TileSizeY, TileSizeX), M /
                                   // TileSizeY, N / TileSizeX)
    auto const tiled_tensor_dst{cute::tiled_divide(
        tensor_dst, block_shape_transposed)}; // ((TileSizeX, TileSizeY), N
                                              // / TileSizeX, M / TileSizeY)
    auto const tiled_tensor_dst_transposed{cute::tiled_divide(
        tensor_dst_transposed, block_shape)}; // ((TileSizeY, TileSizeX), M
                                              // / TileSizeY, N / TileSizeX)

    using ThreadBlockSizeX = cute::Int<32>; // tN
    using ThreadBlockSizeY = cute::Int<8>;  // tM

    CUTE_STATIC_ASSERT(TileSizeX::value % ThreadBlockSizeX::value == 0,
                       "TileSizeX must be divisible by ThreadBlockSizeX");
    CUTE_STATIC_ASSERT(TileSizeY::value % ThreadBlockSizeY::value == 0,
                       "TileSizeY must be divisible by ThreadBlockSizeY");

    constexpr auto thread_block_shape{
        cute::make_shape(ThreadBlockSizeY{}, ThreadBlockSizeX{})};
    constexpr auto thread_block_shape_transposed{
        cute::make_shape(ThreadBlockSizeX{}, ThreadBlockSizeY{})};
    constexpr auto thread_layout{
        cute::make_layout(thread_block_shape, cute::GenRowMajor{})};
    constexpr auto thread_layout_transposed{
        cute::make_layout(thread_block_shape_transposed, cute::GenColMajor{})};

    dim3 const grid_dim{cute::size<2>(tiled_tensor_src),
                        cute::size<1>(tiled_tensor_src)};
    dim3 const thread_dim{ThreadBlockSizeX::value * ThreadBlockSizeY::value};

    using VECTOR_SIZE_X = cute::Int<NUM_VECTOR_ELEMENTS>;
    constexpr auto vector_shape{
        cute::make_shape(cute::Int<1>{}, VECTOR_SIZE_X{})};
    // Copy atom vector layout.
    constexpr auto vector_layout{
        cute::make_layout(vector_shape, cute::GenRowMajor{})};

    matrix_transpose_shared_memory_vectorized<<<grid_dim, thread_dim, 0,
                                                stream>>>(
        tiled_tensor_src, tiled_tensor_dst_transposed,
        shared_memory_layout_src_padded, shared_memory_layout_src_padded,
        thread_layout, thread_layout_transposed, vector_layout);

    return hipGetLastError();
}

template <class SHARED_MEMORY_LAYOUT>
static void
print_shared_memory_bank_ids(SHARED_MEMORY_LAYOUT shared_memory_layout)
{
    // Print the shared memory bank ids.
    for (unsigned int i{0}; i < cute::size<0>(shared_memory_layout); ++i)
    {
        for (unsigned int j{0}; j < cute::size<1>(shared_memory_layout); ++j)
        {
            std::cout << std::setw(2) << shared_memory_layout(i, j) % 32 << " ";
        }
        std::cout << std::endl;
    }
}

constexpr int constexpr_log2(int n)
{
    return ((n < 2) ? 0 : 1 + constexpr_log2(n / 2));
}

template <typename T>
static hipError_t launch_matrix_transpose_shared_memory_swizzled(
    T const* input_matrix, T* output_matrix, unsigned int M, unsigned int N,
    hipStream_t stream)
{
    auto const tensor_shape{cute::make_shape(M, N)};
    auto const tensor_shape_transposed{cute::make_shape(N, M)};

    // Input matrix: row-major M x N matrix.
    auto const global_memory_layout_src{cute::make_layout(
        tensor_shape, cute::GenRowMajor{})}; // (M, N) : (N, 1)
    // Output matrix: row-major N x M matrix.
    auto const global_memory_layout_dst{cute::make_layout(
        tensor_shape_transposed, cute::GenRowMajor{})}; // (N, M) : (M, 1)
    // Same output matrix, but different view: column-major M x N matrix.
    auto const global_memory_layout_dst_transposed{cute::make_layout(
        tensor_shape, cute::GenColMajor{})}; // (M, N) : (1, M)

    auto const tensor_src{cute::make_tensor(cute::make_gmem_ptr(input_matrix),
                                            global_memory_layout_src)};
    auto const tensor_dst{cute::make_tensor(cute::make_gmem_ptr(output_matrix),
                                            global_memory_layout_dst)};
    auto const tensor_dst_transposed{
        cute::make_tensor(cute::make_gmem_ptr(output_matrix),
                          global_memory_layout_dst_transposed)};

    using TileSizeX = cute::Int<64>; // bN
    using TileSizeY = cute::Int<32>; // bM
    constexpr int NUM_BASE_BITS{constexpr_log2(1)};
    constexpr int NUM_MASK_BITS{constexpr_log2(32 * 4 / sizeof(T)) -
                                NUM_BASE_BITS};
    constexpr int NUM_SHIFT_BITS{constexpr_log2(TileSizeX::value) -
                                 NUM_BASE_BITS};

    constexpr auto block_shape{cute::make_shape(TileSizeY{}, TileSizeX{})};
    constexpr auto block_shape_transposed{
        cute::make_shape(TileSizeX{}, TileSizeY{})};

    auto const shared_memory_layout_src{cute::make_layout(
        block_shape, cute::GenRowMajor{})}; // (bM, bN) : (bN, 1)
    auto const shared_memory_layout_dst{cute::make_layout(
        block_shape_transposed, cute::GenRowMajor{})}; // (bN, bM) : (bM, 1)
    auto const shared_memory_layout_dst_transposed{cute::make_layout(
        block_shape, cute::GenColMajor{})}; // (bM, bN) : (1, bM)

    auto const swizzle_src{
        cute::Swizzle<NUM_MASK_BITS, NUM_BASE_BITS, NUM_SHIFT_BITS>{}};
    auto const shared_memory_layout_swizzled_src{
        cute::composition(swizzle_src, shared_memory_layout_src)};

    // Inspect if the swizzling reduces the shared memory bank conflicts.
    // print_shared_memory_bank_ids(shared_memory_layout_swizzled_src);

    auto const tiled_tensor_src{cute::tiled_divide(
        tensor_src, block_shape)}; // ((TileSizeY, TileSizeX), M /
                                   // TileSizeY, N / TileSizeX)
    auto const tiled_tensor_dst{cute::tiled_divide(
        tensor_dst, block_shape_transposed)}; // ((TileSizeX, TileSizeY), N
                                              // / TileSizeX, M / TileSizeY)
    auto const tiled_tensor_dst_transposed{cute::tiled_divide(
        tensor_dst_transposed, block_shape)}; // ((TileSizeY, TileSizeX), M
                                              // / TileSizeY, N / TileSizeX)

    using ThreadBlockSizeX = cute::Int<32>; // tN
    using ThreadBlockSizeY = cute::Int<8>;  // tM

    CUTE_STATIC_ASSERT(TileSizeX::value % ThreadBlockSizeX::value == 0,
                       "TileSizeX must be divisible by ThreadBlockSizeX");
    CUTE_STATIC_ASSERT(TileSizeY::value % ThreadBlockSizeY::value == 0,
                       "TileSizeY must be divisible by ThreadBlockSizeY");

    constexpr auto thread_block_shape{
        cute::make_shape(ThreadBlockSizeY{}, ThreadBlockSizeX{})};
    constexpr auto thread_block_shape_transposed{
        cute::make_shape(ThreadBlockSizeX{}, ThreadBlockSizeY{})};
    constexpr auto thread_layout{
        cute::make_layout(thread_block_shape, cute::GenRowMajor{})};
    constexpr auto thread_layout_transposed{
        cute::make_layout(thread_block_shape_transposed, cute::GenColMajor{})};

    dim3 const grid_dim{cute::size<2>(tiled_tensor_src),
                        cute::size<1>(tiled_tensor_src)};
    dim3 const thread_dim{ThreadBlockSizeX::value * ThreadBlockSizeY::value};

    matrix_transpose_shared_memory<<<grid_dim, thread_dim, 0, stream>>>(
        tiled_tensor_src, tiled_tensor_dst_transposed,
        shared_memory_layout_swizzled_src, shared_memory_layout_swizzled_src,
        thread_layout, thread_layout_transposed);

    return hipGetLastError();
}

template <typename T>
static hipError_t launch_matrix_transpose_shared_memory_vectorized_swizzled(
    T const* input_matrix, T* output_matrix, unsigned int M, unsigned int N,
    hipStream_t stream)
{
    using VectorType = cute::uint128_t;
    static_assert(sizeof(VectorType) % sizeof(T) == 0,
                  "sizeof(VectorType) must be a multiple of sizeof(T)");
    constexpr unsigned int NUM_VECTOR_ELEMENTS{sizeof(VectorType) / sizeof(T)};

    if (N % NUM_VECTOR_ELEMENTS != 0)
    {
        return hipErrorInvalidValue;
    }

    auto const tensor_shape{cute::make_shape(M, N)};
    auto const tensor_shape_transposed{cute::make_shape(N, M)};

    // Input matrix: row-major M x N matrix.
    auto const global_memory_layout_src{cute::make_layout(
        tensor_shape, cute::GenRowMajor{})}; // (M, N) : (N, 1)
    // Output matrix: row-major N x M matrix.
    auto const global_memory_layout_dst{cute::make_layout(
        tensor_shape_transposed, cute::GenRowMajor{})}; // (N, M) : (M, 1)
    // Same output matrix, but different view: column-major M x N matrix.
    auto const global_memory_layout_dst_transposed{cute::make_layout(
        tensor_shape, cute::GenColMajor{})}; // (M, N) : (1, M)

    auto const tensor_src{cute::make_tensor(cute::make_gmem_ptr(input_matrix),
                                            global_memory_layout_src)};
    auto const tensor_dst{cute::make_tensor(cute::make_gmem_ptr(output_matrix),
                                            global_memory_layout_dst)};
    auto const tensor_dst_transposed{
        cute::make_tensor(cute::make_gmem_ptr(output_matrix),
                          global_memory_layout_dst_transposed)};

    using TileSizeX = cute::Int<128>; // bN
    using TileSizeY = cute::Int<32>;  // bM
    constexpr int NUM_BASE_BITS{constexpr_log2(NUM_VECTOR_ELEMENTS)};
    constexpr int NUM_MASK_BITS{constexpr_log2(32 * 4 / sizeof(T)) -
                                NUM_BASE_BITS};
    constexpr int NUM_SHIFT_BITS{constexpr_log2(TileSizeX::value) -
                                 NUM_BASE_BITS};

    constexpr auto block_shape{cute::make_shape(TileSizeY{}, TileSizeX{})};
    constexpr auto block_shape_transposed{
        cute::make_shape(TileSizeX{}, TileSizeY{})};

    auto const shared_memory_layout_src{cute::make_layout(
        block_shape, cute::GenRowMajor{})}; // (bM, bN) : (bN, 1)
    auto const shared_memory_layout_dst{cute::make_layout(
        block_shape_transposed, cute::GenRowMajor{})}; // (bN, bM) : (bM, 1)
    auto const shared_memory_layout_dst_transposed{cute::make_layout(
        block_shape, cute::GenColMajor{})}; // (bM, bN) : (1, bM)

    // Because of the vectorized access, NUM_BASE_BITS cannot be zero.
    // The shared memory bank conflict mitigation can be compromised.
    // Print the shared memory bank ids to see the details.
    auto const swizzle_src{
        cute::Swizzle<NUM_MASK_BITS, NUM_BASE_BITS, NUM_SHIFT_BITS>{}};
    auto const shared_memory_layout_swizzled_src{
        cute::composition(swizzle_src, shared_memory_layout_src)};

    // Inspect if the swizzling reduces the shared memory bank conflicts.
    // print_shared_memory_bank_ids(shared_memory_layout_swizzled_src);

    auto const tiled_tensor_src{cute::tiled_divide(
        tensor_src, block_shape)}; // ((TileSizeY, TileSizeX), M /
                                   // TileSizeY, N / TileSizeX)
    auto const tiled_tensor_dst{cute::tiled_divide(
        tensor_dst, block_shape_transposed)}; // ((TileSizeX, TileSizeY), N
                                              // / TileSizeX, M / TileSizeY)
    auto const tiled_tensor_dst_transposed{cute::tiled_divide(
        tensor_dst_transposed, block_shape)}; // ((TileSizeY, TileSizeX), M
                                              // / TileSizeY, N / TileSizeX)

    using ThreadBlockSizeX = cute::Int<32>; // tN
    using ThreadBlockSizeY = cute::Int<8>;  // tM

    CUTE_STATIC_ASSERT(TileSizeX::value % ThreadBlockSizeX::value == 0,
                       "TileSizeX must be divisible by ThreadBlockSizeX");
    CUTE_STATIC_ASSERT(TileSizeY::value % ThreadBlockSizeY::value == 0,
                       "TileSizeY must be divisible by ThreadBlockSizeY");

    constexpr auto thread_block_shape{
        cute::make_shape(ThreadBlockSizeY{}, ThreadBlockSizeX{})};
    constexpr auto thread_block_shape_transposed{
        cute::make_shape(ThreadBlockSizeX{}, ThreadBlockSizeY{})};
    constexpr auto thread_layout{
        cute::make_layout(thread_block_shape, cute::GenRowMajor{})};
    constexpr auto thread_layout_transposed{
        cute::make_layout(thread_block_shape_transposed, cute::GenColMajor{})};

    using VECTOR_SIZE_X = cute::Int<NUM_VECTOR_ELEMENTS>;
    constexpr auto vector_shape{
        cute::make_shape(cute::Int<1>{}, VECTOR_SIZE_X{})};
    // Copy atom vector layout.
    constexpr auto vector_layout{
        cute::make_layout(vector_shape, cute::GenRowMajor{})};

    dim3 const grid_dim{cute::size<2>(tiled_tensor_src),
                        cute::size<1>(tiled_tensor_src)};
    dim3 const thread_dim{ThreadBlockSizeX::value * ThreadBlockSizeY::value};

    matrix_transpose_shared_memory_vectorized<<<grid_dim, thread_dim, 0,
                                                stream>>>(
        tiled_tensor_src, tiled_tensor_dst_transposed,
        shared_memory_layout_swizzled_src, shared_memory_layout_swizzled_src,
        thread_layout, thread_layout_transposed, vector_layout);

    return hipGetLastError();
}

// Explicit instantiation.
template hipError_t
launch_matrix_transpose_shared_memory_bank_conflict_read<float>(
    float const* input_matrix, float* output_matrix, unsigned int M,
    unsigned int N, hipStream_t stream);
template hipError_t
launch_matrix_transpose_shared_memory_bank_conflict_read<double>(
    double const* input_matrix, double* output_matrix, unsigned int M,
    unsigned int N, hipStream_t stream);

template hipError_t
launch_matrix_transpose_shared_memory_vectorized_bank_conflict_read<float>(
    float const* input_matrix, float* output_matrix, unsigned int M,
    unsigned int N, hipStream_t stream);
template hipError_t
launch_matrix_transpose_shared_memory_vectorized_bank_conflict_read<double>(
    double const* input_matrix, double* output_matrix, unsigned int M,
    unsigned int N, hipStream_t stream);

template hipError_t
launch_matrix_transpose_shared_memory_bank_conflict_write<float>(
    float const* input_matrix, float* output_matrix, unsigned int M,
    unsigned int N, hipStream_t stream);
template hipError_t
launch_matrix_transpose_shared_memory_bank_conflict_write<double>(
    double const* input_matrix, double* output_matrix, unsigned int M,
    unsigned int N, hipStream_t stream);

template hipError_t launch_matrix_transpose_shared_memory_padded<float>(
    float const* input_matrix, float* output_matrix, unsigned int M,
    unsigned int N, hipStream_t stream);
template hipError_t launch_matrix_transpose_shared_memory_padded<double>(
    double const* input_matrix, double* output_matrix, unsigned int M,
    unsigned int N, hipStream_t stream);

template hipError_t
launch_matrix_transpose_shared_memory_vectorized_padded<float>(
    float const* input_matrix, float* output_matrix, unsigned int M,
    unsigned int N, hipStream_t stream);
template hipError_t
launch_matrix_transpose_shared_memory_vectorized_padded<double>(
    double const* input_matrix, double* output_matrix, unsigned int M,
    unsigned int N, hipStream_t stream);

template hipError_t launch_matrix_transpose_shared_memory_swizzled<float>(
    float const* input_matrix, float* output_matrix, unsigned int M,
    unsigned int N, hipStream_t stream);
template hipError_t launch_matrix_transpose_shared_memory_swizzled<double>(
    double const* input_matrix, double* output_matrix, unsigned int M,
    unsigned int N, hipStream_t stream);

template hipError_t
launch_matrix_transpose_shared_memory_vectorized_swizzled<float>(
    float const* input_matrix, float* output_matrix, unsigned int M,
    unsigned int N, hipStream_t stream);
template hipError_t
launch_matrix_transpose_shared_memory_vectorized_swizzled<double>(
    double const* input_matrix, double* output_matrix, unsigned int M,
    unsigned int N, hipStream_t stream);
