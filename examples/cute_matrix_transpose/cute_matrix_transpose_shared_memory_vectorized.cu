#include "hip/hip_runtime.h"
#include <iomanip>
#include <iostream>

#include <hip/hip_runtime.h>

#include <cute/tensor.hpp>
#include <cutlass/array.h>

#include "cute_matrix_transpose.hpp"

template <class TENSOR_SRC, class TENSOR_DST, class SHARED_MEMORY_LAYOUT_SRC,
          class SHARED_MEMORY_LAYOUT_DST, class THREAD_LAYOUT_SRC,
          class THREAD_LAYOUT_DST, class VECTOR_LAYOUT>
static __global__ void
transpose_shared_memory(TENSOR_SRC tensor_src, TENSOR_DST tensor_dst,
                        SHARED_MEMORY_LAYOUT_SRC, SHARED_MEMORY_LAYOUT_DST,
                        THREAD_LAYOUT_SRC, THREAD_LAYOUT_DST, VECTOR_LAYOUT)
{
    using Element = typename TENSOR_SRC::value_type;
    CUTE_STATIC_ASSERT(cute::size(SHARED_MEMORY_LAYOUT_SRC{}) ==
                           cute::size(SHARED_MEMORY_LAYOUT_DST{}),
                       "SHARED_MEMORY_LAYOUT_SRC and SHARED_MEMORY_LAYOUT_DST "
                       "must have the same size.");
    __shared__ Element shared_memory[cute::cosize(SHARED_MEMORY_LAYOUT_SRC{})];

    auto tensor_cache_src{cute::make_tensor(cute::make_smem_ptr(shared_memory),
                                            SHARED_MEMORY_LAYOUT_SRC{})};
    auto tensor_cache_dst{cute::make_tensor(cute::make_smem_ptr(shared_memory),
                                            SHARED_MEMORY_LAYOUT_DST{})};

    auto global_tile_src{tensor_src(cute::make_coord(cute::_, cute::_),
                                    blockIdx.y,
                                    blockIdx.x)}; // (TILE_SIZE_Y, TILE_SIZE_X)
    auto global_tile_dst{tensor_dst(cute::make_coord(cute::_, cute::_),
                                    blockIdx.y,
                                    blockIdx.x)}; // (TILE_SIZE_Y, TILE_SIZE_X)

    auto thread_global_tile_src{cute::local_partition(
        global_tile_src, THREAD_LAYOUT_SRC{},
        threadIdx.x)}; // (THREAD_VALUE_SIZE_Y, THREAD_VALUE_SIZE_X)
    auto thread_global_tile_dst{cute::local_partition(
        global_tile_dst, THREAD_LAYOUT_DST{},
        threadIdx.x)}; // (THREAD_VALUE_SIZE_X, THREAD_VALUE_SIZE_Y)

    auto thread_shared_tile_src{cute::local_partition(
        tensor_cache_src, THREAD_LAYOUT_SRC{},
        threadIdx.x)}; // (THREAD_VALUE_SIZE_Y, THREAD_VALUE_SIZE_X)
    auto thread_shared_tile_dst{cute::local_partition(
        tensor_cache_dst, THREAD_LAYOUT_DST{},
        threadIdx.x)}; // (THREAD_VALUE_SIZE_X, THREAD_VALUE_SIZE_Y)

    using AccessType = cutlass::AlignedArray<Element, cute::size(VECTOR_LAYOUT{})>;
    using Atom = cute::Copy_Atom<cute::UniversalCopy<AccessType>, Element>;
    auto tiled_input_copy{cute::make_tiled_copy(
        Atom{},                       // access size
        THREAD_LAYOUT_SRC{},          // thread layout
        VECTOR_LAYOUT{})};                 // vector layout (e.g. 4x1)
    auto thread_input_copy = tiled_input_copy.get_thread_slice(threadIdx.x);

    auto thread_global_tile_src_vectorized{thread_input_copy.partition_S(global_tile_src)};
    auto thread_shared_tile_src_vectorized{thread_input_copy.partition_D(tensor_cache_src)};
    auto fragment = cute::make_fragment_like(thread_shared_tile_src_vectorized);             // (CopyOp, CopyM, CopyN)

    if (blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0)
    {
        printf("--------------------\n");
        cute::print(tiled_input_copy);
        printf("\n");
        printf("--------------------\n");
        cute::print(thread_input_copy);
        printf("\n");
        printf("--------------------\n");
        cute::print(thread_global_tile_src_vectorized);
        printf("\n");
        printf("--------------------\n");
        cute::print(thread_shared_tile_src_vectorized);
        printf("\n");
        printf("--------------------\n");
        cute::print(fragment);
        printf("\n");
        printf("--------------------\n");
    }

    cute::copy(tiled_input_copy, thread_global_tile_src_vectorized,
                  fragment);
    cute::copy(tiled_input_copy, fragment,
                  thread_shared_tile_src_vectorized);
    cute::cp_async_fence();
    cute::cp_async_wait<0>();
    __syncthreads();
    // cute::copy(thread_shared_tile_dst, thread_global_tile_dst);



    // // Construct a register-backed Tensor with the same shape as each thread's partition
    // // Use make_fragment because the first mode is the instruction-local mode
    // Tensor fragment = make_fragment_like(thr_tile_D);             // (CopyOp, CopyM, CopyN)

    // // Copy from GMEM to RMEM and from RMEM to GMEM
    // copy(tiled_copy, thr_tile_S, fragment);
    // copy(tiled_copy, fragment, thr_tile_D);






    // // A 2D array of tuples that maps (x, y) to (x, y).
    // auto const identity_tensor_src{cute::make_identity_tensor(cute::make_shape(
    //     cute::size<0>(global_tile_src), cute::size<1>(global_tile_src)))};
    // auto const thread_identity_tensor_src{cute::local_partition(
    //     identity_tensor_src, THREAD_LAYOUT_SRC{}, threadIdx.x)};
    // auto predicator_src{cute::make_tensor<bool>(
    //     cute::make_shape(cute::size<0>(thread_global_tile_src),
    //                      cute::size<1>(thread_global_tile_src)))};

    // auto const identity_tensor_dst{cute::make_identity_tensor(cute::make_shape(
    //     cute::size<0>(global_tile_dst), cute::size<1>(global_tile_dst)))};
    // auto const thread_identity_tensor_dst{cute::local_partition(
    //     identity_tensor_dst, THREAD_LAYOUT_DST{}, threadIdx.x)};
    // auto predicator_dst{cute::make_tensor<bool>(
    //     cute::make_shape(cute::size<0>(thread_global_tile_dst),
    //                      cute::size<1>(thread_global_tile_dst)))};

    // auto const num_max_columns{cute::stride<0>(global_tile_src)};
    // auto const num_max_rows{cute::stride<1>(global_tile_dst)};
    // constexpr auto global_tile_columns{cute::size<1>(global_tile_src)};
    // constexpr auto global_tile_rows{cute::size<0>(global_tile_src)};

    // CUTE_UNROLL
    // for (unsigned int i{0}; i < cute::size<0>(predicator_src); ++i)
    // {
    //     CUTE_UNROLL
    //     for (unsigned int j{0}; j < cute::size<1>(predicator_src); ++j)
    //     {
    //         auto const thread_identity{thread_identity_tensor_src(i, j)};
    //         bool const is_row_in_bound{cute::get<0>(thread_identity) +
    //                                        blockIdx.y * global_tile_rows <
    //                                    num_max_rows};
    //         bool const is_column_in_bound{cute::get<1>(thread_identity) +
    //                                           blockIdx.x * global_tile_columns <
    //                                       num_max_columns};
    //         predicator_src(i, j) = is_row_in_bound && is_column_in_bound;
    //     }
    // }

    // CUTE_UNROLL
    // for (unsigned int i{0}; i < cute::size<0>(predicator_dst); ++i)
    // {
    //     CUTE_UNROLL
    //     for (unsigned int j{0}; j < cute::size<1>(predicator_dst); ++j)
    //     {
    //         auto const thread_identity{thread_identity_tensor_dst(i, j)};
    //         bool const is_row_in_bound{cute::get<0>(thread_identity) +
    //                                        blockIdx.y * global_tile_rows <
    //                                    num_max_rows};
    //         bool const is_column_in_bound{cute::get<1>(thread_identity) +
    //                                           blockIdx.x * global_tile_columns <
    //                                       num_max_columns};
    //         predicator_dst(i, j) = is_row_in_bound && is_column_in_bound;
    //     }
    // }

    // cute::copy_if(predicator_src, thread_global_tile_src,
    //               thread_shared_tile_src);
    // cute::cp_async_fence();
    // cute::cp_async_wait<0>();
    // __syncthreads();
    // cute::copy_if(predicator_dst, thread_shared_tile_dst,
    //               thread_global_tile_dst);
}

enum class SharedMemoryBankConflictAccessMode
{
    Read,
    Write
};

template <typename T>
hipError_t launch_transpose_shared_memory_bank_conflict_base(
    T const* input_matrix, T* output_matrix, unsigned int M, unsigned int N,
    SharedMemoryBankConflictAccessMode bank_conflict_access_mode,
    hipStream_t stream)
{
    auto const tensor_shape{cute::make_shape(M, N)};
    auto const tensor_shape_transposed{cute::make_shape(N, M)};

    // Input matrix: row-major M x N matrix.
    auto const global_memory_layout_src{cute::make_layout(
        tensor_shape, cute::GenRowMajor{})}; // (M, N) : (N, 1)
    // Output matrix: row-major N x M matrix.
    auto const global_memory_layout_dst{cute::make_layout(
        tensor_shape_transposed, cute::GenRowMajor{})}; // (N, M) : (M, 1)
    // Same output matrix, but different view: column-major M x N matrix.
    auto const global_memory_layout_dst_transposed{cute::make_layout(
        tensor_shape, cute::GenColMajor{})}; // (M, N) : (1, M)

    auto const tensor_src{cute::make_tensor(cute::make_gmem_ptr(input_matrix),
                                            global_memory_layout_src)};
    auto const tensor_dst{cute::make_tensor(cute::make_gmem_ptr(output_matrix),
                                            global_memory_layout_dst)};
    auto const tensor_dst_transposed{
        cute::make_tensor(cute::make_gmem_ptr(output_matrix),
                          global_memory_layout_dst_transposed)};

    using TILE_SIZE_X = cute::Int<128>; // bN
    using TILE_SIZE_Y = cute::Int<32>; // bM

    constexpr auto block_shape{cute::make_shape(TILE_SIZE_Y{}, TILE_SIZE_X{})};
    constexpr auto block_shape_transposed{
        cute::make_shape(TILE_SIZE_X{}, TILE_SIZE_Y{})};

    auto const shared_memory_layout_src{cute::make_layout(
        block_shape, cute::GenRowMajor{})}; // (bM, bN) : (bN, 1)
    auto const shared_memory_layout_dst{cute::make_layout(
        block_shape_transposed, cute::GenRowMajor{})}; // (bN, bM) : (bM, 1)
    auto const shared_memory_layout_dst_transposed{cute::make_layout(
        block_shape, cute::GenColMajor{})}; // (bM, bN) : (1, bM)

    auto const tiled_tensor_src{cute::tiled_divide(
        tensor_src, block_shape)}; // ((TILE_SIZE_Y, TILE_SIZE_X), M /
                                   // TILE_SIZE_Y, N / TILE_SIZE_X)
    auto const tiled_tensor_dst{cute::tiled_divide(
        tensor_dst, block_shape_transposed)}; // ((TILE_SIZE_X, TILE_SIZE_Y), N
                                              // / TILE_SIZE_X, M / TILE_SIZE_Y)
    auto const tiled_tensor_dst_transposed{cute::tiled_divide(
        tensor_dst_transposed, block_shape)}; // ((TILE_SIZE_Y, TILE_SIZE_X), M
                                              // / TILE_SIZE_Y, N / TILE_SIZE_X)

    using THREAD_BLOCK_SIZE_X = cute::Int<32>; // tN
    using THREAD_BLOCK_SIZE_Y = cute::Int<8>;  // tM

    CUTE_STATIC_ASSERT(TILE_SIZE_X::value % THREAD_BLOCK_SIZE_X::value == 0,
                       "TILE_SIZE_X must be divisible by THREAD_BLOCK_SIZE_X");
    CUTE_STATIC_ASSERT(TILE_SIZE_Y::value % THREAD_BLOCK_SIZE_Y::value == 0,
                       "TILE_SIZE_Y must be divisible by THREAD_BLOCK_SIZE_Y");

    constexpr auto thread_block_shape{
        cute::make_shape(THREAD_BLOCK_SIZE_Y{}, THREAD_BLOCK_SIZE_X{})};
    constexpr auto thread_block_shape_transposed{
        cute::make_shape(THREAD_BLOCK_SIZE_X{}, THREAD_BLOCK_SIZE_Y{})};
    constexpr auto thread_layout{
        cute::make_layout(thread_block_shape, cute::GenRowMajor{})};
    constexpr auto thread_layout_transposed{
        cute::make_layout(thread_block_shape_transposed, cute::GenColMajor{})};
    
    constexpr auto vector_layout{cute::make_layout(cute::make_shape(cute::Int<1>{}, cute::Int<4>{}), cute::GenRowMajor{})};

    dim3 const grid_dim{cute::size<2>(tiled_tensor_src),
                        cute::size<1>(tiled_tensor_src)};
    dim3 const thread_dim{
        cute::size(THREAD_BLOCK_SIZE_X::value * THREAD_BLOCK_SIZE_Y::value)};

    if (bank_conflict_access_mode == SharedMemoryBankConflictAccessMode::Read)
    {
        transpose_shared_memory<<<grid_dim, thread_dim, 0, stream>>>(
            tiled_tensor_src, tiled_tensor_dst_transposed,
            shared_memory_layout_src, shared_memory_layout_src, thread_layout,
            thread_layout_transposed, vector_layout);
    }
    else
    {
        transpose_shared_memory<<<grid_dim, thread_dim, 0, stream>>>(
            tiled_tensor_src, tiled_tensor_dst_transposed,
            shared_memory_layout_dst_transposed,
            shared_memory_layout_dst_transposed, thread_layout,
            thread_layout_transposed, vector_layout);
    }

    return hipGetLastError();
}

template <typename T>
hipError_t launch_transpose_shared_memory_bank_conflict_read_vectorized(
    T const* input_matrix, T* output_matrix, unsigned int M, unsigned int N,
    hipStream_t stream)
{
    return launch_transpose_shared_memory_bank_conflict_base(
        input_matrix, output_matrix, M, N,
        SharedMemoryBankConflictAccessMode::Read, stream);
}

template <typename T>
hipError_t launch_transpose_shared_memory_bank_conflict_write_vectorized(
    T const* input_matrix, T* output_matrix, unsigned int M, unsigned int N,
    hipStream_t stream)
{
    return launch_transpose_shared_memory_bank_conflict_base(
        input_matrix, output_matrix, M, N,
        SharedMemoryBankConflictAccessMode::Write, stream);
}

// Explicit instantiation.
template hipError_t launch_transpose_shared_memory_bank_conflict_read_vectorized<float>(
    float const* input_matrix, float* output_matrix, unsigned int M,
    unsigned int N, hipStream_t stream);
template hipError_t launch_transpose_shared_memory_bank_conflict_read_vectorized<double>(
    double const* input_matrix, double* output_matrix, unsigned int M,
    unsigned int N, hipStream_t stream);

// template hipError_t launch_transpose_shared_memory_bank_conflict_write<float>(
//     float const* input_matrix, float* output_matrix, unsigned int M,
//     unsigned int N, hipStream_t stream);
// template hipError_t launch_transpose_shared_memory_bank_conflict_write<double>(
//     double const* input_matrix, double* output_matrix, unsigned int M,
//     unsigned int N, hipStream_t stream);
